#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "MatrixOperations.cuh"
#include "Utilities.cuh"
#include <ctime>

using namespace std;


// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    //err = hipDeviceSynchronize();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}

void printDeviceProperties() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf(" -Device name: %s\n", prop.name);
        printf(" -Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf(" -Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf(" -Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf(" -Max number of threads per block: %d\n\n",
               prop.maxThreadsPerBlock);
        printf(" -Max number of blocks: %dx%dx%d\n\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf(" -Total number of multiprocessors: %d\n\n",
               prop.multiProcessorCount);


    }
}

short* flattenMatrix(short** matrix, int cols, int rows){
  short* flat = new short[rows*cols];
  int currentPixel = 0;
  for(int r = 0; r < rows; ++r){
    for(int c = 0; c < cols; ++c){
      flat[currentPixel] = matrix[r][c];
      ++currentPixel;
    }
  }
  cout<<"Matrix is flattened."<<endl;
  return flat;
}

short** expandMatrix(short* flattened, int cols, int rows){
  short** expanded = new short*[rows];
  int currentPixel = 0;
  for(int r = 0; r < rows; ++r){
    short* currentRow = new short[cols];
    for(int c = 0; c < cols; ++c){
      currentRow[c] = flattened[currentPixel];
      ++currentPixel;
    }
    expanded[r] = currentRow;
  }
  cout<<"Array is now a Matrix."<<endl;
  return expanded;
}


short** incrementMatrix(short alter, short** matrix, int cols, int rows){
  for(int r = 0; r < rows; ++r){
    for(int c = 0; c < cols; ++c){
      matrix[r][c] += alter;
    }
  }
  return matrix;
}


short** hostTranspose(short** matrix, int rows, int cols){
  short** transposable = new short*[rows];
  for(int row = 0; row < rows; ++row){
    transposable[row] = new short[cols];
    for(int col = 0; col < cols; ++col){
      transposable[row][col] = matrix[col][row];
    }
    //cout<<"Timepoint "<<row<<" trasposed..."<<endl;

  }

  return transposable;
}

__global__ void transposeShortMatrix(short* flatOrigin, short* flatTransposed, long Nrows, long Ncols){

  long globalID = blockIdx.x * blockDim.x + threadIdx.x;
  long pixel = globalID;
  long stride = gridDim.x * blockDim.x;
  long flatLength = Nrows * Ncols;
  long row = 0;
  long col = 0;
  short currentPixelIntensity = 0;
  while(pixel < flatLength){
    row = pixel/Ncols;
    col = pixel - Ncols*row;
    flatTransposed[pixel] = flatOrigin[row + Nrows*col];
    pixel += stride;
  }

}

int main(){

  time_t timer = time(nullptr);
  printDeviceProperties();

  int numTimePoints = 512;
  int rows  = 2048;
  const int columns = 1024;
  short** testMatrix = new short*[rows];
  for(int i = 0; i < rows; ++i){
    testMatrix[i] = new short[columns];
    for(int c = 0; c < columns; ++c){
      testMatrix[i][c] = c;
    }
  }

  cout<<"Done filling test array at "<<difftime(time(nullptr), timer)<<" second"<<endl;
  short** timePointArray = new short*[numTimePoints];
  for(int i = 0; i < numTimePoints; ++i){

    timePointArray[i] = flattenMatrix(incrementMatrix(1, testMatrix, columns, rows), columns, rows);
  }
  cout<<"Done filling timepoint vector at "<<difftime(time(nullptr), timer)<<" second"<<endl;



  bool transposed = false;
  int Nrows = 0;
  int Ncols = 0;
  short* flattenedFull = flattenMatrix(timePointArray, rows*columns, numTimePoints);//Nrows and Ncols are switched here
  cout<<"Original Array has been flattened"<<endl;
  short* flatTransposed = new short[rows*columns*numTimePoints];//might not be used
  if(rows*columns >= numTimePoints){
    transposed = true;
    Nrows = rows*columns;
    Ncols = numTimePoints;
    //int** transposedMatrix = new int*[Nrows];
    //for(int i = 0; i < Nrows; ++i){
    //  testMatrix[i] = new int[columns];
    //  for(int c = 0; c < Ncols; ++c){
    //    testMatrix[i][c] = 0;
    //  }
    //}
    cout<<"Transpose initiation complete at "<<difftime(time(nullptr), timer)<<" second"<<endl;

    time_t transposeTimer = time(nullptr);

    short** transposedMatrix = hostTranspose(timePointArray, Nrows, Ncols);
    if(transposedMatrix[0] != timePointArray[0] && transposedMatrix[1][0] == timePointArray[0][1]){
      cout<<"SUCCESS IN TRANSPOSITION IN>>>"<<difftime(time(nullptr), transposeTimer)<<" second"<<endl;
    }
    else{
      cout<<"FAILURE IN TRANSPOSITION"<<endl;

      exit(1);
    }


    //transposition
    short* fullDevice;
    short* transDevice;


    CudaSafeCall(hipMalloc((void**)&fullDevice, Nrows*Ncols*sizeof(short)));
    CudaSafeCall(hipMalloc((void**)&transDevice, Nrows*Ncols*sizeof(short)));
    CudaSafeCall(hipMemcpy(fullDevice, flattenedFull, Nrows*Ncols*sizeof(short), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(transDevice, flatTransposed, Nrows*Ncols*sizeof(short), hipMemcpyHostToDevice));


    unsigned int numBlocks = 2147483647;
    while(Nrows * Ncols < numBlocks){
      numBlocks--;
    }
    long startIndex = 0;
    long endIndex = numBlocks;
    transposeTimer = time(nullptr);
    cout<<"LENGTH OF FLAT MATRIX = "<<Nrows * Ncols<<endl;
    cout<<"MATRIX DIM = "<<Nrows <<"x"<<Ncols<<endl;
    transposeShortMatrix<<<numBlocks,1>>>(fullDevice, transDevice, Nrows, Ncols);
    CudaCheckError();

    CudaSafeCall(hipMemcpy(flatTransposed, transDevice, Nrows*Ncols*sizeof(short), hipMemcpyDeviceToHost));
    short** pixelsByTimePoints = expandMatrix(flatTransposed, Ncols, Nrows);


    if(pixelsByTimePoints[0] != timePointArray[0] && pixelsByTimePoints[1][0] == timePointArray[0][1]){
        cout<<"SUCCESS IN TRANSPOSITION KERNEL IN>>>"<<difftime(time(nullptr), transposeTimer)<<" second"<<endl;
    }
    else{
        cout<<"FAILURE IN TRANSPOSITION KERNEL"<<endl;
        exit(1);

    }
  }
  else{
    Nrows = numTimePoints;
    Ncols = rows*columns;
  }

  short* flatSVDMatrix;

  if(transposed){//use flatTransposed
    flatSVDMatrix = flatTransposed;
  }
  else{//use flattenedFull
    flatSVDMatrix = flattenedFull;
  }

  /*
  cout<<"Printing last timepoint:"<<endl;
  for(int i = 0; i < rows*columns; ++i){
    if(i%columns == 0){
      cout<<endl;
    }
    cout<<timepoints[numTimePoints - 1][i]<<" ";
  }
  cout<<endl;
  */


  //SVD time!!!!
  // --- gesvd only supports Nrows >= Ncols
  // --- column major memory ordering
  //thanks OrangOwlSolutions
  //https://github.com/OrangeOwlSolutions/Linear-Algebra/blob/master/SVD/SVD.cu

  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  //WE NEED TO TRANSPOSE MATRIX
  //THIS IS BECAUSE WE CURRENTLY HAVE TIMEPOINTS AS ROWS
  //now: matrix = numTimePointsX(rows*columns)
  //needs to be (rows*columns)*numTimePoints
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

/*
  // --- cuSOLVE input/output parameters/arrays
  int work_size = 0;
  int *devInfo;
  gpuErrchk(hipMalloc(&devInfo,sizeof(int)));

  // --- CUDA solver initialization
  hipsolverHandle_t solver_handle;
  hipsolverDnCreate(&solver_handle);

  // --- Setting the host, Nrows x Ncols matrix
  double *h_A = (double *)malloc(Nrows * Ncols * sizeof(double));
  for(int j = 0; j < Nrows; j++){
    for(int i = 0; i < Ncols; i++){
      h_A[j + i*Nrows] = (i + j*j) * sqrt((double)(i + j));
    }
  }

  // --- Setting the device matrix and moving the host matrix to the device
  double *d_A;
  gpuErrchk(hipMalloc(&d_A, Nrows * Ncols * sizeof(double)));
  gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

  // --- host side SVD results space
  double *h_U = (double *)malloc(Nrows * Nrows     * sizeof(double));
  double *h_V = (double *)malloc(Ncols * Ncols     * sizeof(double));
  double *h_S = (double *)malloc(min(Nrows, Ncols) * sizeof(double));

  // --- device side SVD workspace and matrices
  double *d_U;
  gpuErrchk(hipMalloc(&d_U,	Nrows * Nrows     * sizeof(double)));
  double *d_V;
  gpuErrchk(hipMalloc(&d_V,	Ncols * Ncols	  * sizeof(double)));
  double *d_S;
  gpuErrchk(hipMalloc(&d_S,	min(Nrows, Ncols) * sizeof(double)));

  // --- CUDA SVD initialization
  cusolveSafeCall(hipsolverDnDgesvd_bufferSize(solver_handle, Nrows, Ncols, &work_size));
  double *work;
  gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));

  // --- CUDA SVD execution
  cusolveSafeCall(hipsolverDnDgesvd(solver_handle, 'A', 'A', Nrows, Ncols, d_A, Nrows, d_S, d_U, Nrows, d_V, Ncols, work, work_size, NULL, devInfo));
  int devInfo_h = 0;
  gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  if (devInfo_h != 0){
    std::cout	<< "Unsuccessful SVD execution\n\n";
  }

  // --- Moving the results from device to host
  gpuErrchk(hipMemcpy(h_S, d_S, min(Nrows, Ncols) * sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h_U, d_U, Nrows * Nrows     * sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h_V, d_V, Ncols * Ncols     * sizeof(double), hipMemcpyDeviceToHost));

  std::cout << "Singular values\n";
  for(int i = 0; i < min(Nrows, Ncols); i++){
    std::cout << "d_S["<<i<<"] = " << std::setprecision(15) << h_S[i] << std::endl;
  }
  std::cout << "\nLeft singular vectors - For y = A * x, the columns of U span the space of y\n";
  for(int j = 0; j < Nrows; j++) {
    printf("\n");
    for(int i = 0; i < Nrows; i++)
      printf("U[%i,%i]=%f\n",i,j,h_U[j*Nrows + i]);
  }

  std::cout << "\nRight singular vectors - For y = A * x, the columns of V span the space of x\n";
  for(int i = 0; i < Ncols; i++) {
    printf("\n");
    for(int j = 0; j < Ncols; j++){
      printf("V[%i,%i]=%f\n",i,j,h_V[j*Ncols + i]);
    }
  }

  hipsolverDnDestroy(solver_handle);
*/


  return 0;

}
