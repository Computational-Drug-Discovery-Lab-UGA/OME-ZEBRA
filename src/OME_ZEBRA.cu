#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <vector>
#include <inttypes.h>
#include "tiffio.h"
#include <fstream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
using namespace std;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    //err = hipDeviceSynchronize();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}





void printDeviceProperties();
string createFourCharInt(int i);
void printArray(uint32 * array, uint32 width);
uint32* extractMartrices(TIFF* tif, string fileName);
uint32* extractMartrices(TIFF* tif, string fileName, int currentTimePoint);
uint32* extractMartrices(TIFF* tif);
vector<uint32> flattenMatrix(vector<uint32*> matrix, int cols, int rows);
uint32** hostTranspose(uint32** matrix, int rows, int cols);
__global__ void transposeuint32Matrix(uint32* flatOrigin, uint32* flatTransposed, long Nrows, long Ncols);
uint32 findMin(uint32* flatMatrix, int size);
__global__ void calcCa(uint32* flatMatrix, uint32 min, uint32 max, long size);
__global__ void fillTestMatrix(uint32* flatMatrix, long size);

int main(int argc, char *argv[]) {

    if(argc != 3) {
      cout << "Usage: ./exe <file> <# of time points>";
      return 1;
    }
    else {

      vector<uint32*> flattenedTimePoints;
      string baseName = argv[1];
      int numTimePoints = atoi(argv[2]);
      if(numTimePoints == 0){
        cout<<"ERROR INVALID TIMEPOINTS"<<endl;
        exit(-1);
      }
      bool allTifsAreGood = true;
      uint32 numColumns;
      uint32 numRows;
      string currentTif;
      for(int i = 0; i < numTimePoints; ++i){

        currentTif = "data/registeredOMEs/" + baseName + "/" +
        baseName + ".ome" + createFourCharInt(i) + ".tif";

        TIFF* tif = TIFFOpen(currentTif.c_str(), "r");

        if (tif) {
          if(i == 0){
            TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &numColumns);
            TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &numRows);
          }
          uint32 tempCol;
          uint32 tempRow;
          cout<<endl<<currentTif<<" IS OPENED\n"<<endl;
          TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &tempCol);
          TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &tempRow);
          if(numRows != tempRow || numColumns != tempCol){
            cout<<"ERROR NOT ALL TIFFS ARE THE SAME LENGTH"<<endl;
            exit(-1);
          }

          uint32* flatMatrix = new uint32[numRows*numColumns];
          if(i == 0){
            flatMatrix = extractMartrices(tif, baseName);
          }
          else{
            flatMatrix = extractMartrices(tif);
          }
          flattenedTimePoints.push_back(flatMatrix);
          TIFFClose(tif);

        }
        else{
          allTifsAreGood = false;
          break;
        }
      }
      if (allTifsAreGood) {



          cout<<"Creating key"<<endl;
          int NNormal = numTimePoints;
          int MNormal = (numRows*numColumns);

          bool* key = new bool[MNormal];
          for (int i = 0; i < MNormal; i++) {
            key[i] = false;
          }


          uint32* temp;
          uint32 min = 4294967295;
          uint32 max = 0;
          temp = new uint32[MNormal*NNormal];
          int indexOfTemp = 0;
          int nonZeroCounter = 0;
          uint32* rowArray = new uint32[numColumns];
          int rowArrayIndex = 0;
          int lastGoodIndex = 0;
          bool allRealRows = false;
          for(unsigned i=0; i < MNormal; i++) {
            allRealRows = false;
            nonZeroCounter = 0;
            rowArrayIndex = 0;
            for(unsigned j=0; j < NNormal; j++) {
              if (flattenedTimePoints[j][i] != 0){
                nonZeroCounter++;
                if(flattenedTimePoints[j][i] < min) min = flattenedTimePoints[j][i];
                if(flattenedTimePoints[j][i] > max) max = flattenedTimePoints[j][i];

              }
              rowArray[rowArrayIndex] = flattenedTimePoints[j][i];
              rowArrayIndex++;
            }

            if (nonZeroCounter != 0) {
              for (int k = 0; k < NNormal; k++) {
                temp[indexOfTemp] = rowArray[k];
                rowArray[k] = 0;
                indexOfTemp++;
              }
              lastGoodIndex++;
              key[i] = true;
              allRealRows = true;
            }
          }
          if(allRealRows){
            cout<<"key created but all pixels have at least 1 non-zero value"<<endl;
          }
          else{

            cout<<"key created and all temporal zero pixels have that been removed are indicated by 0 in the key"<<endl;
          }
          cout << lastGoodIndex << endl;
          long minimizedSize = lastGoodIndex*512;
          uint32* actualArray = new uint32[minimizedSize];
          cout << "loading arrays based on key" << endl;
          for (long i = 0; i < minimizedSize; i++) {

            actualArray[i] = temp[i];

          }
          dim3 grid = {1,1,1};
          dim3 block = {1,1,1};
          if(65535 > minimizedSize){
            grid.x = minimizedSize;
          }
          else if(65535*1024 > minimizedSize){
            grid.x = 65535;
            block.x = 1024;
            while(block.x*grid.x > minimizedSize){
              block.x--;
            }
          }
          else{
            grid.x = 65535;
            block.x = 1024;
            while(grid.x*grid.y*block.x < minimizedSize){
              grid.y++;
            }
          }
          cout<<"prepare for calcCa cuda kernel with min = "<<min<<",max = "<<max<<endl;
          uint32* actualArrayDevice;
          CudaSafeCall(hipMalloc((void**)&actualArrayDevice,minimizedSize*sizeof(uint32)));
          CudaSafeCall(hipMemcpy(actualArrayDevice,actualArray, minimizedSize*sizeof(uint32), hipMemcpyHostToDevice));
          calcCa<<<grid,block>>>(actualArrayDevice, min, max, minimizedSize);
          CudaCheckError();
          CudaSafeCall(hipMemcpy(actualArray,actualArrayDevice, minimizedSize*sizeof(uint32), hipMemcpyDeviceToHost));
          CudaSafeCall(hipFree(actualArrayDevice));
          cout<<"calcCa has completed applying offset"<<endl;

          cout << "Dumping to File" << endl;

          ofstream myfile ("data/NNMF.nmf");
          if (myfile.is_open()) {
            for(long count = 0; count < ((lastGoodIndex) * 512); count++){

              if ((count + 1) % 512 == 0) {

                 myfile << actualArray[count] << "\n" ;

              }
              else {

                myfile << actualArray[count] << " " ;

              }
            }
            myfile.close();
          }

          ofstream mykeyfile ("data/key.csv");
          if (mykeyfile.is_open()) {
            for(long i = 0; i < MNormal; i++){

               mykeyfile << key[i] << "\n" ;

             }

           }
            mykeyfile.close();
            cout<<"NNMF.csv created successfuly"<<endl;

          }
          else{
            cout<<"ERROR OPENING TIFF IN THIS DIRECTORY"<<endl;
            exit(-1);
          }

      }


      return 0;

    }



//method implementations


void printDeviceProperties(){
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf(" -Device name: %s\n", prop.name);
        printf(" -Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf(" -Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf(" -Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf(" -Max number of threads per block: %d\n\n",
               prop.maxThreadsPerBlock);
        printf(" -Max number of blocks: %dx%dx%d\n\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf(" -Total number of multiprocessors: %d\n\n",
               prop.multiProcessorCount);


    }
}
string createFourCharInt(int i){
  string strInt;
  if(i < 10){
    strInt = "000" + to_string(i);
  }
  else if(i < 100){
    strInt = "00" + to_string(i);
  }
  else if(i < 1000){
    strInt = "0" + to_string(i);
  }
  else{
    strInt = to_string(i);
  }
  return strInt;
}
void printArray(uint32 * array, uint32 width){
    uint32 i;
    for (i=0;i<width;i++){
      printf("%u ", array[i]);
    }
    cout<<endl;
}

uint32* extractMartrices(TIFF* tif, string fileName){
  string newtiff = fileName.substr(0, fileName.length() - 8) + "_TP1.tif";
  TIFF* firstTimePoint = TIFFOpen(newtiff.c_str(), "w");
  if(firstTimePoint){
    tdata_t buf;
    uint32 config;

    uint32 height, width, photo;
    short samplesPerPixel, bitsPerSample;
    tsize_t scanLineSize;

    TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &width);
    TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &height);
    TIFFGetField(tif, TIFFTAG_SAMPLESPERPIXEL, &samplesPerPixel);
    TIFFGetField(tif, TIFFTAG_BITSPERSAMPLE, &bitsPerSample);
    TIFFGetField(tif, TIFFTAG_PHOTOMETRIC, &photo);

    uint32* currentTimePoint = new uint32[width*height];

    TIFFSetField(firstTimePoint, TIFFTAG_IMAGEWIDTH, width);
    TIFFSetField(firstTimePoint, TIFFTAG_IMAGELENGTH, height);
    TIFFSetField(firstTimePoint, TIFFTAG_SAMPLESPERPIXEL, samplesPerPixel);
    TIFFSetField(firstTimePoint, TIFFTAG_BITSPERSAMPLE,bitsPerSample);
    TIFFSetField(firstTimePoint, TIFFTAG_PHOTOMETRIC, photo);
    cout<<"\nTIMEPOINT 1 .tif info:"<<endl;
    printf("width = %d\nheight = %d\nsamplesPerPixel = %d\nbitsPerSample = %d\n\n",width,height,samplesPerPixel,bitsPerSample);
    scanLineSize = TIFFScanlineSize(tif);
    buf = _TIFFmalloc(scanLineSize);
    cout<<"TIFF SCANLINE SIZE IS "<<scanLineSize<<" bits"<<endl;
    //printf("Height,Width = %u,%u -> scanLineSize = %d bytes\n", height, width,TIFFScanlineSize(tif));
    for (uint32 row = 0; row < height; row++){
      if(TIFFReadScanline(tif, buf, row, 0) != -1){
        memcpy(&currentTimePoint[row*width], buf, scanLineSize);
        if(TIFFWriteScanline(firstTimePoint, buf, row, 0) == -1){
          cout<<"ERROR WRITING SCANLINE"<<endl;
          exit(-1);
        }
      }
      else{
        cout<<"ERROR READING SCANLINE"<<endl;
        exit(-1);
      }
    }
    TIFFClose(firstTimePoint);
    _TIFFfree(buf);
    return currentTimePoint;
  }
  else{
    cout<<"COULD NOT CREATE FIRST TIMEPOINT TIFF"<<endl;
    exit(-1);
  }
}
uint32* extractMartrices(TIFF* tif, string fileName, int currentTimePoint){
  string newtiff = fileName.substr(0, fileName.length() - 8) + "_TP" + to_string(currentTimePoint) + ".tif";
  TIFF* currentDir = TIFFOpen(newtiff.c_str(), "w");
  if(currentDir){
    tdata_t buf;
    uint32 config;

    uint32 height, width, photo;
    short samplesPerPixel, bitsPerSample;
    tsize_t scanLineSize;

    TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &width);
    TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &height);
    TIFFGetField(tif, TIFFTAG_SAMPLESPERPIXEL, &samplesPerPixel);
    TIFFGetField(tif, TIFFTAG_BITSPERSAMPLE, &bitsPerSample);
    TIFFGetField(tif, TIFFTAG_PHOTOMETRIC, &photo);

    uint32* timePoint = new uint32[width*height];

    TIFFSetField(currentDir, TIFFTAG_IMAGEWIDTH, width);
    TIFFSetField(currentDir, TIFFTAG_IMAGELENGTH, height);
    TIFFSetField(currentDir, TIFFTAG_SAMPLESPERPIXEL, samplesPerPixel);
    TIFFSetField(currentDir, TIFFTAG_BITSPERSAMPLE,bitsPerSample);
    TIFFSetField(currentDir, TIFFTAG_PHOTOMETRIC, photo);
    cout<<"\nTIMEPOINT "<<currentTimePoint<<" .tif info:"<<endl;
    printf("width = %d\nheight = %d\nsamplesPerPixel = %d\nbitsPerSample = %d\n\n",width,height,samplesPerPixel,bitsPerSample);
    scanLineSize = TIFFScanlineSize(tif);
    buf = _TIFFmalloc(scanLineSize);
    cout<<"TIFF SCANLINE SIZE IS "<<scanLineSize<<" bits"<<endl;
    //printf("Height,Width = %u,%u -> scanLineSize = %d bytes\n", height, width,TIFFScanlineSize(tif));
    for (uint32 row = 0; row < height; row++){
      if(TIFFReadScanline(tif, buf, row, 0) != -1){
        memcpy(&timePoint[row*width], buf, scanLineSize);
        if(TIFFWriteScanline(currentDir, buf, row, 0) == -1){
          cout<<"ERROR WRITING SCANLINE"<<endl;
          exit(-1);
        }
      }
      else{
        cout<<"ERROR READING SCANLINE"<<endl;
        exit(-1);
      }
    }
    TIFFClose(currentDir);
    _TIFFfree(buf);
    return timePoint;
  }
  else{
    cout<<"COULD NOT CREATE FIRST TIMEPOINT TIFF"<<endl;
    exit(-1);
  }
}
uint32* extractMartrices(TIFF* tif){

  uint32 height,width;
  tdata_t buf;
  uint32 config;
  vector<uint32*> currentPlane;
  tsize_t scanLineSize;

  TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &width);
  TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &height);

  uint32* currentTimePoint = new uint32[width*height];
  scanLineSize = TIFFScanlineSize(tif);
  buf = _TIFFmalloc(scanLineSize);

  //printf("Height,Width = %u,%u -> scanLineSize = %d bytes\n", height, width,TIFFScanlineSize(tif));
  for (uint32 row = 0; row < height; row++){
    if(TIFFReadScanline(tif, buf, row, 0) != -1){
      memcpy(&currentTimePoint[row*width], buf, scanLineSize);
    }
    else{
      cout<<"ERROR READING SCANLINE"<<endl;
      exit(-1);
    }

  }
  _TIFFfree(buf);
  return currentTimePoint;
}

uint32** hostTranspose(uint32** matrix, int rows, int cols){
  uint32** transposable = new uint32*[rows];
  for(int row = 0; row < rows; ++row){
    transposable[row] = new uint32[cols];
    for(int col = 0; col < cols; ++col){
      transposable[row][col] = matrix[col][row];
    }
    //cout<<"Timepoint "<<row<<" trasposed..."<<endl;

  }

  return transposable;
}

__global__ void transposeuint32Matrix(uint32* flatOrigin, uint32* flatTransposed, long Nrows, long Ncols){

  long globalID = blockIdx.x * blockDim.x + threadIdx.x;
  long pixel = globalID;
  long stride = gridDim.x * blockDim.x;
  long flatLength = Nrows * Ncols;
  long row = 0;
  long col = 0;
  uint32 currentPixelIntensity = 0;
  while(pixel < flatLength){
    row = pixel/Ncols;
    col = pixel - Ncols*row;
    flatTransposed[pixel] = flatOrigin[row + Nrows*col];
    pixel += stride;
  }

}

vector<uint32> flattenMatrix(vector<uint32*> matrix, int cols, int rows){
  vector<uint32> flat;
  for(int r = 0; r < rows; ++r){
    for(int c = 0; c < cols; ++c){
      flat.push_back(matrix[r][c]);
    }
  }
  //cout<<"Matrix is flattened."<<endl;
  return flat;
}
uint32 findMin(uint32* flatMatrix, int size){
  uint32 currentMin = 0;
  for(int i = 0; i < size; ++i){
    if(currentMin > flatMatrix[i]){
      currentMin = flatMatrix[i];
    }
  }
  return currentMin;
}
__global__ void calcCa(uint32* flatMatrix, uint32 min, uint32 max, long size){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  int globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  long currentIndex = globalID;
  uint32 caConc = 0;
  uint32 currentIntensity = 0;
  uint32 firingRate = 0;
  while(currentIndex < size){
    currentIntensity = flatMatrix[globalID];
    caConc = 3.16227766e-7*((currentIntensity - min)/(max - currentIntensity));
    flatMatrix[globalID] = caConc + 1;
    currentIndex += stride;
  }
}
__global__ void fillTestMatrix(uint32* flatMatrix, long size){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  int globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  long currentIndex = globalID;
  hiprandState state;
  while(currentIndex < size){
    hiprand_init(clock64(), currentIndex, 0, &state);
    flatMatrix[currentIndex] = hiprand_uniform(&state);
    currentIndex += stride;
  }
}
