#include "common_includes.h"
#include "io_util.cuh"
#include "cuda_zebra.cuh"

int main(int argc, char *argv[]) {
  if(argc < 2 || argc > 3){
    std::cout << "Usage: ./exe <directory of timepoint tifs>";
    exit(-1);
  }
  unsigned int k;
  if(argc == 3) k = std::stoi(argv[2]);
  else{
    k = 2;
  }
  std::string baseDirectory = argv[1];
  unsigned int width = 0;
  unsigned int height = 0;
  unsigned int numTimePoints = 0;
  std::string baseName = "";
  uint32* tifVideo = readTiffVideo(baseDirectory, width, height, numTimePoints, baseName);
  float* normVideo = executeNormalization(tifVideo, width*height*numTimePoints);
  delete[] tifVideo;
  unsigned long numPixelsWithValues = 0;
  bool* key = generateKey(width*height, numTimePoints, normVideo, numPixelsWithValues);
  float* minimizedVideo;
  if(numPixelsWithValues != height*width){
    minimizedVideo = minimizeVideo(height*width, numPixelsWithValues, numTimePoints, normVideo, key);
    delete[] normVideo;
  }
  else{
    minimizedVideo = normVideo;
  }
  float* W;
  float* H;
  //W & H are not currently and neither are any c files
  //NOTE minimized video is deleted in performNNMF
  performNNMF(W, H, minimizedVideo, k, height*width, numTimePoints, baseDirectory);
  createVisualization(baseDirectory,k, width, height, numTimePoints, W, H, key, baseName);
  hipDeviceReset();
  return 0;
}
