#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <vector>
#include <inttypes.h>
#include "tiffio.h"
#include <fstream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <cfloat>
using namespace std;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    //err = hipDeviceSynchronize();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}

/*
METHOD DECLARATIONS
*/

void printDeviceProperties();
string createFourCharInt(int i);
void printArray(uint32 * array, uint32 width);
uint32* extractMartrices(TIFF* tif, string fileName);
uint32* extractMartrices(TIFF* tif);
vector<uint32> flattenMatrix(vector<uint32*> matrix, int cols, int rows);
uint32** hostTranspose(uint32** matrix, int rows, int cols);
__global__ void transposeuint32Matrix(uint32* flatOrigin, uint32* flatTransposed, long Nrows, long Ncols);
uint32 findMin(uint32* flatMatrix, int size);
__global__ void calcCa(uint32* flatMatrix, float* calcium, uint32 min, long size);
__global__ void calcFiringRate(float* frMatrix, long size, int numTimePoints);
__global__ void calcFiringRateExpanded(float* frMatrix, long size, int numTimePoints);
__global__ void fillTestMatrix(uint32* flatMatrix, long size);
void transposeArray(vector<uint32*> inputArray, int n, int m, uint32 * outputArray, uint32 & min, uint32 & max);

/*
MAIN
*/

int main(int argc, char *argv[]) {

    if(argc != 3) {
      cout << "Usage: ./exe <file> <# of time points>";
      return 1;
    }
    else {

      vector<uint32*> flattenedTimePoints;
      string baseName = argv[1];
      int numTimePoints = atoi(argv[2]);
      if(numTimePoints == 0){
        cout<<"ERROR INVALID TIMEPOINTS"<<endl;
        exit(-1);
      }
      bool allTifsAreGood = true;
      uint32 numColumns;
      uint32 numRows;
      string currentTif;
      for(int i = 0; i < numTimePoints; ++i){

        currentTif = "data/registeredOMEs/" + baseName + "/" +
        baseName + ".ome" + createFourCharInt(i) + ".tif";

        TIFF* tif = TIFFOpen(currentTif.c_str(), "r");

        if (tif) {
          if(i == 0){
            TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &numColumns);
            TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &numRows);
          }
          uint32 tempCol;
          uint32 tempRow;
          cout<<currentTif<<" IS OPENED"<<endl;
          TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &tempCol);
          TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &tempRow);
          if(numRows != tempRow || numColumns != tempCol){
            cout<<"ERROR NOT ALL TIFFS ARE THE SAME LENGTH"<<endl;
            exit(-1);
          }

          uint32* flatMatrix = new uint32[numRows*numColumns];
          flatMatrix = extractMartrices(tif);
          flattenedTimePoints.push_back(flatMatrix);
          TIFFClose(tif);

        }
        else{
          allTifsAreGood = false;
          break;
        }
      }
      if (allTifsAreGood) {

          int NNormal = numTimePoints;
          int MNormal = (numRows*numColumns);

          cout<<"flattening"<<endl;

          uint32 min = UINT32_MAX;
          uint32 max = 0;
          uint32* temp = new uint32[MNormal*NNormal];
          int indexOfTemp = 0;
          int nonZeroCounter = 0;
          uint32* rowArray = new uint32[numColumns];
          int rowArrayIndex = 0;
          for(unsigned i=0; i < MNormal; i++) {

            nonZeroCounter = 0;
            rowArrayIndex = 0;
            for(unsigned j=0; j < NNormal; j++) {
              if (flattenedTimePoints[j][i] != 0){
                nonZeroCounter++;
                if(flattenedTimePoints[j][i] < min) min = flattenedTimePoints[j][i];
                if(flattenedTimePoints[j][i] > max) max = flattenedTimePoints[j][i];
              }

              rowArray[rowArrayIndex] = flattenedTimePoints[j][i];
              rowArrayIndex++;
            }
            for (int k = 0; k < NNormal; k++) {

              temp[indexOfTemp] = rowArray[k];
              rowArray[k] = 0;
              indexOfTemp++;

            }
          }
          //need to delete all flattenedTimePoints arrays
          delete[] rowArray;

          uint32* actualArray = new uint32[MNormal*NNormal];
          float* firingRateArray = new float[MNormal*NNormal];
          cout << "loading arrays" << endl;

          for (long i = 0; i < MNormal*NNormal; i++) {
            //firingRateArray[i] = 0.0f;
            actualArray[i] = temp[i];

          }

          dim3 grid = {1,1,1};
          dim3 block = {1,1,1};

          if(65535 > MNormal*NNormal){
            grid.x = MNormal*NNormal;
          }
          else if(65535*1024 > MNormal*NNormal){
            grid.x = 65535;
            block.x = 1024;
            while(block.x*grid.x > MNormal*NNormal){
              block.x--;
            }
          }
          else{
            grid.x = 65535;
            block.x = 1024;
            while(grid.x*grid.y*block.x < MNormal*NNormal){
              grid.y++;
            }
          }
          cout<<"prepare for calcCa cuda kernel with min = "<<min<<",max = "<<max<<endl;
          float* firingRateArrayDevice;
          uint32* actualArrayDevice;
          CudaSafeCall(hipMalloc((void**)&actualArrayDevice,MNormal*NNormal*sizeof(uint32)));
          CudaSafeCall(hipMalloc((void**)&firingRateArrayDevice,MNormal*NNormal*sizeof(float)));
          CudaSafeCall(hipMemcpy(actualArrayDevice,actualArray, MNormal*NNormal*sizeof(uint32), hipMemcpyHostToDevice));
          CudaSafeCall(hipMemcpy(firingRateArrayDevice,firingRateArray, MNormal*NNormal*sizeof(float), hipMemcpyHostToDevice));
          calcCa<<<grid,block>>>(actualArrayDevice, firingRateArrayDevice, min,MNormal*NNormal);
          CudaCheckError();
          CudaSafeCall(hipMemcpy(firingRateArray,firingRateArrayDevice, MNormal*NNormal*sizeof(float), hipMemcpyDeviceToHost));
          for(int i = 0; i < MNormal*NNormal; ++i){
            if(!std::isfinite(firingRateArray[i])){
              cout<<"ERROR NON FINITE CALCIUM CONCENTRATION "<<firingRateArray[i]<<endl;
              exit(-1);
            }
            if(firingRateArray[i] < 0.0f){
              cout<<"ERROR NEGATIVE CALCIUM CONCENTRATION "<<firingRateArray[i]<<endl;
              exit(-1);
            }

          }
          cout<<"Executing firing rate cuda kernel"<<endl;
          calcFiringRate<<<grid,block>>>(firingRateArrayDevice, MNormal*NNormal, numTimePoints);
          CudaSafeCall(hipMemcpy(firingRateArray,firingRateArrayDevice, MNormal*NNormal*sizeof(float), hipMemcpyDeviceToHost));
          CudaSafeCall(hipFree(actualArrayDevice));
          CudaSafeCall(hipFree(firingRateArrayDevice));
          delete[] actualArray;
          cout<<"calcCa has completed applying offset"<<endl;

          float* tempCalc = new float[MNormal*NNormal];
          indexOfTemp = 0;
          int lastGoodIndex = 0;

          float *newRowArray = new float[NNormal];
          float calcMin = FLT_MAX;
          float calcMax = 0;
          cout<<"Creating key"<<endl;

          bool* key = new bool[MNormal];
          for (int i = 0; i < MNormal; i++) {

            key[i] = false;


          }

          for(unsigned i=0; i < MNormal; i++) {

            nonZeroCounter = 0;
            for(unsigned j=0; j < NNormal; j++) {

              if (firingRateArray[(NNormal*i) + j] != 0.0f){
                nonZeroCounter++;
              }
              if(!std::isfinite(firingRateArray[(NNormal*i) + j])){
                cout<<"ERROR NON FINITE NUMBER "<<firingRateArray[(NNormal*i) + j]<<endl;
                exit(-1);
              }
              if(firingRateArray[(NNormal*i) + j] < 0.0f){
                cout<<"ERROR NEGATIVE FIRIING RATE"<<endl;
                exit(-1);
              }
              newRowArray[j] = firingRateArray[(NNormal*i) + j];
            }
            // if (nonZeroCounter != 0) {
            //
            //   for (int k = 0; k < NNormal; k++) {
            //     if(newRowArray[k] < calcMin) calcMin = newRowArray[k];
            //     if(newRowArray[k] > calcMax) calcMax = newRowArray[k];
            //     tempCalc[indexOfTemp] = newRowArray[k];
            //     newRowArray[k] = 0.0f;
            //     indexOfTemp++;
            //     key[i] = true;
            //
            //   }
            //
            //   lastGoodIndex++;
            //
            // }
            // else{
            //   cout<<"EMPTY ROW FOR PIXEL "<<i<<endl;
            // }

            for (int k = 0; k < NNormal; k++) {
              if(newRowArray[k] < calcMin) calcMin = newRowArray[k];
              if(newRowArray[k] > calcMax) calcMax = newRowArray[k];
              tempCalc[indexOfTemp] = newRowArray[k];
              newRowArray[k] = 0.0f;
              indexOfTemp++;
              key[i] = true;

            }

            lastGoodIndex++;

          }
          cout << lastGoodIndex << endl;
          if(lastGoodIndex == NNormal - 1){
            cout<<"KEY CREATED BUT ALL PIXELS HAVE ATLEAST 1 NONZERO VALUE"<<endl;
          }
          cout << "MAX = "<<calcMax<<" AND MIN = "<<calcMin<<endl;

          delete[] firingRateArray;
          cout << "Dumping to File" << endl;

          ofstream myfile ("data/NNMF.nmf");
          if (myfile.is_open()) {
            for(int i = 0; i < (lastGoodIndex)*NNormal; i++){

              if ((i + 1) % 512 == 0) {

                myfile << tempCalc[i] << "\n" ;
                //myfile << (tempCalc[i] + calcMin*-1)/calcMax << "\n" ;

              }
              else {

                myfile << tempCalc[i] << " " ;
                //myfile << (tempCalc[i] + calcMin*-1)/calcMax << " " ;

              }
            }
            myfile.close();
          }

          cout << "done" << endl;

          ofstream mykeyfile ("data/key.csv");
          if (mykeyfile.is_open()) {
            for(long i = 0; i < MNormal; i++){

               mykeyfile << key[i] << "\n" ;

             }

           }
            mykeyfile.close();
            cout<<"NNMF.nmf created successfuly"<<endl;

          }
          else{
            cout<<"ERROR OPENING TIFF IN THIS DIRECTORY"<<endl;
            exit(-1);
          }

      }


      return 0;

    }

/*
METHOD IMPLEMENTATIONS
*/

void printDeviceProperties(){
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf(" -Device name: %s\n", prop.name);
        printf(" -Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf(" -Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf(" -Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf(" -Max number of threads per block: %d\n\n",
               prop.maxThreadsPerBlock);
        printf(" -Max number of blocks: %dx%dx%d\n\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf(" -Total number of multiprocessors: %d\n\n",

               prop.multiProcessorCount);


    }
}

string createFourCharInt(int i){
  string strInt;
  if(i < 10){
    strInt = "000" + to_string(i);
  }
  else if(i < 100){
    strInt = "00" + to_string(i);
  }
  else if(i < 1000){
    strInt = "0" + to_string(i);
  }
  else{
    strInt = to_string(i);
  }
  return strInt;
}

void printArray(uint32 * array, uint32 width){
    uint32 i;
    for (i=0;i<width;i++){
      printf("%u ", array[i]);
    }
    cout<<endl;
}

uint32* extractMartrices(TIFF* tif, string fileName){
  TIFF* firstTimePoint = TIFFOpen(fileName.c_str(), "w");
  if(firstTimePoint){
    tdata_t buf;

    uint32 height, width, photo;
    short samplesPerPixel, bitsPerSample;
    tsize_t scanLineSize;

    TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &width);
    TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &height);
    TIFFGetField(tif, TIFFTAG_SAMPLESPERPIXEL, &samplesPerPixel);
    TIFFGetField(tif, TIFFTAG_BITSPERSAMPLE, &bitsPerSample);
    TIFFGetField(tif, TIFFTAG_PHOTOMETRIC, &photo);

    uint32* currentTimePoint = new uint32[width*height];

    TIFFSetField(firstTimePoint, TIFFTAG_IMAGEWIDTH, width);
    TIFFSetField(firstTimePoint, TIFFTAG_IMAGELENGTH, height);
    TIFFSetField(firstTimePoint, TIFFTAG_SAMPLESPERPIXEL, samplesPerPixel);
    TIFFSetField(firstTimePoint, TIFFTAG_BITSPERSAMPLE,bitsPerSample);
    TIFFSetField(firstTimePoint, TIFFTAG_PHOTOMETRIC, photo);
    cout<<"\nTIMEPOINT 1 .tif info:"<<endl;
    printf("width = %d\nheight = %d\nsamplesPerPixel = %d\nbitsPerSample = %d\n\n",width,height,samplesPerPixel,bitsPerSample);
    scanLineSize = TIFFScanlineSize(tif);
    buf = _TIFFmalloc(scanLineSize);
    cout<<"TIFF SCANLINE SIZE IS "<<scanLineSize<<" bits"<<endl;
    //printf("Height,Width = %u,%u -> scanLineSize = %d bytes\n", height, width,TIFFScanlineSize(tif));
    for (uint32 row = 0; row < height; row++){
      if(TIFFReadScanline(tif, buf, row, 0) != -1){
        memcpy(&currentTimePoint[row*width], buf, scanLineSize);
        if(TIFFWriteScanline(firstTimePoint, buf, row, 0) == -1){
          cout<<"ERROR WRITING SCANLINE"<<endl;
          exit(-1);
        }
      }
      else{
        cout<<"ERROR READING SCANLINE"<<endl;
        exit(-1);
      }
    }
    TIFFClose(firstTimePoint);
    _TIFFfree(buf);
    return currentTimePoint;
  }
  else{
    cout<<"COULD NOT CREATE FIRST TIMEPOINT TIFF"<<endl;
    exit(-1);
  }
}

uint32* extractMartrices(TIFF* tif){

  uint32 height,width;
  tdata_t buf;

  vector<uint32*> currentPlane;
  tsize_t scanLineSize;

  TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &width);
  TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &height);

  uint32* currentTimePoint = new uint32[width*height];
  scanLineSize = TIFFScanlineSize(tif);
  buf = _TIFFmalloc(scanLineSize);

  //printf("Height,Width = %u,%u -> scanLineSize = %d bytes\n", height, width,TIFFScanlineSize(tif));
  for (uint32 row = 0; row < height; row++){
    if(TIFFReadScanline(tif, buf, row, 0) != -1){
      memcpy(&currentTimePoint[row*width], buf, scanLineSize);
    }
    else{
      cout<<"ERROR READING SCANLINE"<<endl;
      exit(-1);
    }

  }
  _TIFFfree(buf);
  return currentTimePoint;
}

uint32** hostTranspose(uint32** matrix, int rows, int cols){
  uint32** transposable = new uint32*[rows];
  for(int row = 0; row < rows; ++row){
    transposable[row] = new uint32[cols];
    for(int col = 0; col < cols; ++col){
      transposable[row][col] = matrix[col][row];
    }
    //cout<<"Timepoint "<<row<<" trasposed..."<<endl;

  }

  return transposable;
}

__global__ void transposeuint32Matrix(uint32* flatOrigin, uint32* flatTransposed, long Nrows, long Ncols){

  long globalID = blockIdx.x * blockDim.x + threadIdx.x;
  long pixel = globalID;
  long stride = gridDim.x * blockDim.x;
  long flatLength = Nrows * Ncols;
  long row = 0;
  long col = 0;
  while(pixel < flatLength){
    row = pixel/Ncols;
    col = pixel - Ncols*row;
    flatTransposed[pixel] = flatOrigin[row + Nrows*col];
    pixel += stride;
  }

}

vector<uint32> flattenMatrix(vector<uint32*> matrix, int cols, int rows){
  vector<uint32> flat;
  for(int r = 0; r < rows; ++r){
    for(int c = 0; c < cols; ++c){
      flat.push_back(matrix[r][c]);
    }
  }
  //cout<<"Matrix is flattened."<<endl;
  return flat;
}

uint32 findMin(uint32* flatMatrix, int size){
  uint32 currentMin = 0;
  for(int i = 0; i < size; ++i){
    if(currentMin > flatMatrix[i]){
      currentMin = flatMatrix[i];
    }
  }
  return currentMin;
}

__global__ void calcCa(uint32* flatMatrix, float* calcium, uint32 min, long size){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float caConc = 0;
  float numerator = 0;
  float denominator = 0;
  float currentValue = 0;
  float dmin =  static_cast<float>(min);
  while(globalID < size){
    if(flatMatrix[globalID] != 0){
      currentValue = static_cast<float>(flatMatrix[globalID]) - dmin;
      numerator = 460*currentValue;
      denominator = (5.5*dmin) - currentValue;
      caConc = numerator/denominator;
    }
    calcium[globalID] = caConc;
    globalID += stride;
  }
}

__global__ void calcFiringRate(float* frMatrix, long size, int numTimePoints){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float caConc = 0.0f;
  float nextCaConc = 0.0f;
  float firingRate = 0.0f;
  float tau = 0.15;
  float expValue = exp(0.0416777/tau);
  float expValuem1 = expm1(0.0416777/tau);
  float multiplier = 1/(tau*250.0f);//250 is in nm
  float numerator = 0.0f;
  int currentTimePoint = globalID % numTimePoints;
  int currentPixel = globalID/numTimePoints;
  while(globalID < size && currentTimePoint < numTimePoints - 1){
    firingRate = 0.0f;
    caConc = frMatrix[globalID];
    nextCaConc = frMatrix[globalID + 1];
    if(nextCaConc != 0.0f){//this will cause firing rate to be 0
      numerator = (nextCaConc*expValue) - caConc;
      if(numerator < 0){//currently these values will be set to 0
        printf("ERROR resulting in negative number %.9f => %.9f, %.9f, TP %d, P %d \n",numerator,caConc, nextCaConc, currentTimePoint, currentPixel);
      }
      else{
        firingRate = multiplier*numerator/expValuem1;
      }
    }
    frMatrix[globalID] = firingRate;
    if(currentTimePoint == numTimePoints - 2){//not sure this is what we want to do
      frMatrix[globalID + 1] = firingRate;
      return;
    }
    globalID += stride;
  }
}

//not implemented
__global__ void calcFiringRateExpanded(float* frMatrix, long size, int numTimePoints){

}

__global__ void fillTestMatrix(uint32* flatMatrix, long size){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  int globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  long currentIndex = globalID;
  hiprandState state;
  while(currentIndex < size){
    hiprand_init(clock64(), currentIndex, 0, &state);
    flatMatrix[currentIndex] = hiprand_uniform(&state);
    currentIndex += stride;
  }

}

void transposeArray(vector<uint32*> inputArray, int n, int m, uint32 * outputArray, uint32 & min, uint32 & max) {

  int outputArrayIndex = 0;

  for(unsigned i=0; i < m; i++) {

    for(unsigned j=0; j < n; j++) {

      if(inputArray[j][i] < min) {

       min = inputArray[j][i];

      }

      if(inputArray[j][i] > max) {

         max = inputArray[j][i];

      }

      outputArray[outputArrayIndex] = inputArray[j][i];
      outputArrayIndex++;

    }

  }

}

void updateHeightMatrix(float* heightMatrix, float* widthMatrix,
  float* uMatrix, float* sMatrix, float* vtMatrix, float* newHeightMatrix,
  int numPixels, int numTime, int numSingularValues) {

    float* widthMatrixTransposedDevice;
    float* uMatrixDevice;
    float* tempSquareMatrixDevice;

    CudaSafeCall(hipMalloc((void**)&widthMatrixTransposedDevice, numPixels * numSingularValues
      * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&uMatrixDevice, numPixels * numSingularValues
      * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&tempSquareMatrixDevice, numSingularValues
      * numSingularValues * sizeof(float)));

    float* widthMatrixTransposed = new float[numPixels * numSingularValues];

    for (int i = 0; i < numPixels; i++) {

      for (int j = 0; j < numSingularValues; j++) {

        widthMatrixTransposed[j * numPixels + i] = widthMatrix[i * numSingularValues + j]

      }

    }

    CudaSafeCall(hipMemcpy(widthMatrixTransposedDevice, widthMatrixTransposed, numPixels
      * numSingularValues * sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(uMatrixDevice, uMatrix, numPixels * numSingularValues
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(widthMatrixDevice, uMatrixDevice, tempSquareMatrixDevice, //TODO setup grid and block size
      numSingularValues, numPixels, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(uMatrixDevice));

    float* sMatrixDevice;
    float* tempSquareMatrix2Device;

    CudaSafeCall(hipMalloc((void**)&sMatrixDevice, numSingularValues
      * numSingularValues * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&tempSquareMatrix2Device, numSingularValues
      * numSingularValues * sizeof(float)));

    CudaSafeCall(hipMemcpy(sMatrixDevice, sMatrix, numSingularValues * numSingularValues
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(tempSquareMatrixDevice, sMatrixDevice, tempSquareMatrix2Device, //TODO setup grid and block size
      numSingularValues, numSingularValues, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(sMatrixDevice));

    float* vtMatrixDevice;
    float* numeratorDevice;

    CudaSafeCall(hipMalloc((void**)&vtMatrixDevice, numSingularValues
      * numTime * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&numeratorDevice, numSingularValues
      * numTime * sizeof(float)));

    CudaSafeCall(hipMemcpy(vtMatrixDevice, vtMatrix, numSingularValues * numTime
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(tempSquareMatrix2Device, vtMatrixDevice, numeratorDevice, //TODO setup grid and block size
      numSingularValues, numSingularValues, numTime);

    CudaCheckError();

    CudaSafeCall(hipFree(tempSquareMatrix2Device));
    CudaSafeCall(hipFree(vtMatrixDevice));

    float* widthMatrixDevice;

    CudaSafeCall(hipMalloc((void**)&widthMatrixDevice, numPixels
      * numSingularValues * sizeof(float)));

    CudaSafeCall(hipMemcpy(widthMatrixDevice, widthMatrix, numPixels * numSingularValues
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(widthMatrixTransposedDevice, widthMatrixDevice, tempSquareMatrixDevice, //TODO setup grid and block size
      numSingularValues, numPixels, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(widthMatrixTransposed));
    CudaSafeCall(hipFree(widthMatrixDevice));

    float* heightMatrixDevice;
    float* denominatorDevice;

    CudaSafeCall(hipMalloc((void**)&heightMatrixDevice, numSingularValues
      * numTime * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&newHeightMatrixDevice, numSingularValues
      * numTime * sizeof(float)));

    CudaSafeCall(hipMemcpy(heightMatrixDevice, heightMatrix, numSingularValues *
      numTime * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(tempSquareMatrixDevice, heightMatrixDevice, //TODO setup grid and block size
      denominatorDevice, numSingularValues, numSingularValues, numTime);

    CudaCheckError();

    CudaSafeCall(hipFree(tempSquareMatrixDevice));

    applyScalar<<<grid,block>>>(heightMatrixDevice, numeratorDevice, //TODO setup grid and block size
      denominatorDevice, numSingularValues, numTime);

    CudaCheckError();

    CudaSafeCall(hipMemcpy(heightMatrix, heightMatrixDevice, numSingularValues*
      * numTime * sizeof(float), hipMemcpyDeviceToHost));

    CudaCheckError();

    CudaSafeCall(hipFree(heightMatrixDevice));
    CudaSafeCall(hipFree(numeratorDevice));
    CudaSafeCall(hipFree(denominatorDevice));

    delete[] widthMatrixTransposed;

  }

void updateWidthMatrix(float* heightMatrix, float* widthMatrix,
  float* uMatrix, float* sMatrix, float* vtMatrix, float* newHeightMatrix,
  int numPixels, int numTime, int numSingularValues) {

    float* heightMatrixTransposedDevice;
    float* vtMatrixDevice;
    float* tempSquareMatrixDevice;

    CudaSafeCall(hipMalloc((void**)&heightMatrixTransposedDevice, numSingularValues
      * numTime * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&vtMatrixDevice, numSingularValues * numTime
      * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&tempSquareMatrixDevice, numSingularValues
      * numSingularValues * sizeof(float)));

    float* heightMatrixTransposed = new float[numSingularValues * numTime];

    for (int i = 0; i < numSingularValues; i++) {

      for (int j = 0; j < numTime; j++) {

        heightMatrixTransposed[j * numSingularValues + i] = widthMatrix[i * numTime + j]

      }

    }

    CudaSafeCall(hipMemcpy(heightMatrixTransposedDevice, heighMatrixTransposed,
      numSingularValues * numTime * sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(vtMatrixDevice, vtMatrix, numSingularValues * numTime
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(vtMatrixDevice, heightMatrixTransposedDevice,
      tempSquareMatrixDevice, numSingularValues, numTime, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(vtMatrixDevice));

    float* sMatrixDevice;
    float* tempSquareMatrix2Device;

    CudaSafeCall(hipMalloc((void**)&sMatrixDevice, numSingularValues
      * numSingularValues * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&tempSquareMatrix2Device, numSingularValues
      * numSingularValues * sizeof(float)));

    CudaSafeCall(hipMemcpy(sMatrixDevice, sMatrix, numSingularValues
      * numSingularValues * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(sMatrixDevice, tempSquareMatrixDevice,
      tempSquareMatrix2Device, numPixels, numSingularValues, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(sMatrixDevice));

    float* uMatrixDevice;
    float* numeratorDevice;

    CudaSafeCall(hipMalloc((void**)&uMatrixDevice, numPixels * numSingularValues
      * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&numeratorDevice, numPixels
      * numSingularValues * sizeof(float)));

    CudaSafeCall(hipMemcpy(uMatrixDevice, uMatrix, numPixels * numSingularValues
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(uMatrixDevice, tempSquareMatrix2Device,
      numeratorDevice, numPixels, numSingularValues, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(uMatrixDevice));
    CudaSafeCall(hipFree(tempSquareMatrix2Device));

    float* heightMatrixDevice;

    CudaSafeCall(hipMalloc((void**)&heightMatrix, numSingularValues * numTime
      * sizeof(float)));

    CudaSafeCall(hipMemcpy(heightMatrixDevice, heightMatrix, numSingularValues
      * numTime * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(heightMatrixDevice, heightMatrixTransposedDevice,
      tempSquareMatrixDevice, numSingularValues, numTime, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(heightMatrixTransposedDevice));
    CudaSafeCall(hipFree(heightMatrixDevice));

    float* widthMatrixDevice;
    float* denominatorDevice;

    CudaSafeCall(hipMalloc((void**)&widthMatrixDevice, numPixels * numSingularValues
      * sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&denominatorDevice, numPixels * numSingularValues
      * sizeof(float)));

    CudaSafeCall(hipMemcpy(widthMatrixDevice, widthMatrix, numPixels * numSingularValues
      * sizeof(float), hipMemcpyHostToDevice));

    multiplyMatrices<<<grid,block>>>(widthMatrixDevice, tempSquareMatrixDevice,
      denominatorDevice, numPixels, numSingularValues, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipFree(tempSquareMatrixDevice));

    applyScalar<<<grid,block>>>(widthMatrixDevice, numeratorDevice, //TODO setup grid and block size
      denominatorDevice, numPixels, numSingularValues);

    CudaCheckError();

    CudaSafeCall(hipMemcpy(widthMatrix, widthMatrixDevice, numPixels *
      * numSingularValues * sizeof(float), hipMemcpyDeviceToHost));

    CudaCheckError();

    CudaSafeCall(hipFree(widthMatrixDevice));
    CudaSafeCall(hipFree(numeratorDevice));
    CudaSafeCall(hipFree(denominatorDevice));

    delete[] heightMatrixTransposed;

}

__global__ void multiplyMatrices(float* matrixA, float* matrixB, float* matrixC, int diffDimA,
   int comDim, int diffDimB) {

     int blockID = blockIdx.y * gridDim.x + blockIdx.x;
     int globalID = blockID * blockDim.x + threadIdx.x;
     long currentIndex = globalID;

     if (currentIndex < (diffDimA * diffDimB)) {

       int iIndex = currentIndex / diffDimB;
       int jIndex = currentIndex % diffDimB;

       int sum = 0;

       for (int k = 0; k < comDim; k++) {

         sum = sum + (matrixA[iIndex * comDim + k] * matrixB[k * diffDimB + jIndex])

       }

       matrixC[iIndex * diffDimB + jIndex] = sum;

     }

   }

__global__ void applyScalar(float* targetMatrix, float* numerator, float* denominator,
    int numRows, int numCols) {

    int blockID = blockIdx.y * gridDim.x + blockIdx.x;
    int globalID = blockID * blockDim.x + threadIdx.x;
    long currentIndex = globalID;

    if (currentIndex < (diffDimA * diffDimB)) {

      int iIndex = currentIndex / diffDimB;
      int jIndex = currentIndex % diffDimB;

      targetMatrix[iIndex * numCols + jIndex] = targetMatrix[iIndex * numCols + jIndex]
        * (numerator[iIndex * numCols + jIndex] / denominator[iIndex * numCols + jIndex]);

    }

  }
