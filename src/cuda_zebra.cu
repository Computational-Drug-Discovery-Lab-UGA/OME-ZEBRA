#include "hip/hip_runtime.h"
#include "cuda_zebra.cuh"

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  // err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

__global__ void findMinMax(uint32* mtx, unsigned long size, uint32* min, uint32* max){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  __shared__ uint32 bmax;
  __shared__ uint32 bmin;
  bmax = 0;
  bmin = UINT32_MAX;
  __syncthreads();
  if(globalID < size){
    uint32 value = mtx[globalID];
    if(value != 0){
      atomicMax(&bmax, value);
      atomicMin(&bmin, value);
    }
  }
  __syncthreads();
  if(threadIdx.x == 0){
    atomicMax(max, bmax);
    atomicMin(min, bmin);
  }
}
__global__ void normalize(uint32 *mtx, float *normals, uint32* min, uint32* max, unsigned long size) {
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float currentValue = 0;
  float dmin = static_cast<float>(*min);
  float dmax = static_cast<float>(*max);
  while(globalID < size){
    if (mtx[globalID] != 0) {
      currentValue = static_cast<float>(mtx[globalID]) - dmin;
      currentValue /= (dmax - dmin);
    }
    normals[globalID] = currentValue;
    normals[globalID] = 1.0f / (1.0f + expf((-10.0f * currentValue) + 7.5));
    //printf("%f\n",normals[globalID]);
    globalID += stride;
  }
}
__global__ void generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, bool* key){
  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numPixels){
    __shared__ bool hasNonZero;
    hasNonZero = false;
    __syncthreads();
    for(int tp = threadIdx.x; tp < numTimePoints; tp += blockDim.x){
      if(hasNonZero) return;
      if(mtx[blockID*numTimePoints + tp] != 0.0f){
        key[blockID] = true;
        hasNonZero = true;
        return;
      }
    }
    __syncthreads();
    if(!hasNonZero){
      key[blockID] = false;
      return;
    }
  }
}
__global__ void randInitMatrix(unsigned long size, float* mtx){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  if(globalID < size){
    mtx[globalID] = ((float)(clock64()%1000))/1000.0f;
    if(mtx[globalID] == 0.0f) mtx[globalID] += 2e-30;
  }
}
__global__ void multiplyMatrices(float *matrixA, float *matrixB, float *matrixC,
                                 long diffDimA, long comDim, long diffDimB){

  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  long currentIndex = globalID;

  if(currentIndex < (diffDimA * diffDimB)){

    long iIndex = currentIndex / diffDimB;
    long jIndex = currentIndex % diffDimB;

    float sum = 0;

    for(int k = 0; k < comDim; k++){

      sum += (matrixA[iIndex * comDim + k] * matrixB[k * diffDimB + jIndex]);
    }

    matrixC[iIndex * diffDimB + jIndex] = sum;
  }
}

void getFlatGridBlock(unsigned long size, dim3 &grid, dim3 &block) {
  if(2147483647 > size){
    grid.x = size;
  }
  else if((unsigned long) 2147483647 * 1024 > size){
    grid.x = 2147483647;
    block.x = 1024;
    while(block.x * grid.x > size){
      block.x--;
    }
    block.x++;
  }
  else{
    grid.x = 65535;
    block.x = 1024;
    grid.y = 1;
    while(grid.x * grid.y * block.x < size){
      grid.y++;
    }
  }
}
void getGrid(unsigned long size, dim3 &grid, int blockSize) {
  if(2147483647 > size){
    grid.x = size;
  }
  else{
    grid.x = 65535;
    grid.y = 1;
    while(grid.x * grid.y * grid.y < size){
      grid.y++;
    }
  }
}
float* executeNormalization(uint32* mtx, unsigned long size){
  uint32 max = 0;
  uint32 min = UINT32_MAX;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(size, grid, block);

  float* norm = new float[size];
  uint32* maxd;
  uint32* mind;
  uint32* matrixDevice;
  float* normDevice;
  CudaSafeCall(hipMalloc((void**)&maxd, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&mind, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&matrixDevice, size*sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&normDevice, size*sizeof(float)));
  CudaSafeCall(hipMemcpy(maxd, &max, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(mind, &min, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, size*sizeof(uint32), hipMemcpyHostToDevice));

  std::cout<<"searching for max and min"<<std::endl;
  findMinMax<<<grid,block>>>(matrixDevice, size, mind, maxd);
  hipDeviceSynchronize();
  CudaCheckError();
  std::cout<<"executing normalization"<<std::endl;
  normalize<<<grid,block>>>(matrixDevice, normDevice, mind, maxd, size);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(&max, maxd, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(&min, mind, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(norm, normDevice, size*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(maxd));
  CudaSafeCall(hipFree(mind));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(normDevice));
  printf("whole video - (uint32) min = %d, max = %d\n",min,max);
  return norm;

}
bool* generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, unsigned long &numPixelsWithValues){
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  block.x = (numTimePoints < 1024) ? numTimePoints : 1024;
  getGrid(numPixels, grid, block.x);

  bool* key = new bool[numPixels];

  float* matrixDevice;
  bool* keyDevice;

  CudaSafeCall(hipMalloc((void**)&matrixDevice, numPixels*numTimePoints*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&keyDevice, numPixels*sizeof(float)));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, numPixels*numTimePoints*sizeof(float), hipMemcpyHostToDevice));
  std::cout<<"generating key to eradicate pixels that are always 0 = ";

  generateKey<<<grid,block>>>(numPixels, numTimePoints, matrixDevice, keyDevice);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(key, keyDevice, numPixels*sizeof(bool), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(keyDevice));
  for(int p = 0; p < numPixels; ++p){
    if(key[p]) ++numPixelsWithValues;
  }
  std::cout<<numPixels - numPixelsWithValues<<std::endl;

  return key;

}
float* minimizeVideo(unsigned long numPixels, unsigned long numPixelsWithValues, unsigned int numTimePoints, float* mtx, bool* key){
  std::cout<<"minimizing video due existence of all 0 rows"<<std::endl;
  float* minimizedVideo = new float[numPixelsWithValues*numTimePoints];
  int currentPixel = 0;
  for(int p = 0; p < numPixels; ++p){
    if(key[p]){
      memcpy(&minimizedVideo[currentPixel*numTimePoints], mtx + p*numTimePoints, numTimePoints*sizeof(float));
      ++currentPixel;
    }
  }
  return minimizedVideo;
}

void performNNMF(float* &W, float* &H, float* V, unsigned int k, unsigned long numPixels, unsigned int numTimePoints){
  float* dW;
  float* dH;

  CudaSafeCall(hipMalloc((void**)&dW, numPixels*k*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&dH, k*numTimePoints*sizeof(float)));
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(numPixels*k, grid, block);
  randInitMatrix<<<grid,block>>>(numPixels*k, dW);
  CudaCheckError();
  grid = {1,1,1};
  block = {1,1,1};
  getFlatGridBlock(k*numTimePoints, grid, block);
  randInitMatrix<<<grid,block>>>(k*numTimePoints, dH);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(W, dW, numPixels*k*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(H, dH, k*numTimePoints*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(dW));
  CudaSafeCall(hipFree(dH));

  clock_t nnmfTimer;
  nnmfTimer = clock();
  std::cout<<"starting nnmf"<<std::endl;
  printf("%f,%f\n",W[0],H[0]);

  /*DO NMF*/


  printf("nnmf took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);
}
