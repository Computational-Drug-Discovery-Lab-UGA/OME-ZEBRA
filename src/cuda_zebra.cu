#include "hip/hip_runtime.h"
#include "cuda_zebra.cuh"

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  // err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

__device__ __forceinline__ int floatToOrderedInt(float floatVal){
 int intVal = __float_as_int( floatVal );
 return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
}
__device__ __forceinline__ float orderedIntToFloat(int intVal){
 return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF);
}

__global__ void ensurePositivity(float* mtx, unsigned long size, int* globalPlaceHolder){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  if(globalID < size){
    atomicMin(globalPlaceHolder, floatToOrderedInt(mtx[globalID]));
    hipDeviceSynchronize();
    mtx[globalID] -= (orderedIntToFloat(*globalPlaceHolder) - 0.1);
  }
}
__global__ void findMinMax(uint32* mtx, unsigned long size, uint32* min, uint32* max){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  __shared__ uint32 bmax;
  __shared__ uint32 bmin;
  bmax = 0;
  bmin = UINT32_MAX;
  __syncthreads();
  if(globalID < size){
    uint32 value = mtx[globalID];
    if(value != 0){
      atomicMax(&bmax, value);
      atomicMin(&bmin, value);
    }
  }
  __syncthreads();
  if(threadIdx.x == 0){
    atomicMax(max, bmax);
    atomicMin(min, bmin);
  }
}
__global__ void normalize(uint32 *mtx, float *normals, uint32* min, uint32* max, unsigned long size) {
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float currentValue = 0;
  float dmin = static_cast<float>(*min);
  float dmax = static_cast<float>(*max);
  while(globalID < size){
    if (mtx[globalID] != 0) {
      currentValue = static_cast<float>(mtx[globalID]) - dmin;
      currentValue /= (dmax - dmin);
    }
    normals[globalID] = currentValue;
    normals[globalID] = 1.0f / (1.0f + expf((-10.0f * currentValue) + 7.5));
    //printf("%f\n",normals[globalID]);
    globalID += stride;
  }
}
__global__ void floatToUINT32(float *mtx, float min, float max, unsigned long size) {
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float currentValue = 0;
  float regMin = min;
  float regMax = max;
  float maxUINT32 = UINT32_MAX;
  while(globalID < size){
    if (mtx[globalID] != 0) {
      currentValue = mtx[globalID] - regMin;
      currentValue /= (regMax - regMin);
    }
    mtx[globalID] = (currentValue*maxUINT32);
    globalID += stride;
  }
}
__global__ void generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, bool* key){
  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numPixels){
    __shared__ bool hasNonZero;
    hasNonZero = false;
    __syncthreads();
    for(int tp = threadIdx.x; tp < numTimePoints; tp += blockDim.x){
      if(hasNonZero) return;
      if(mtx[blockID*numTimePoints + tp] != 0.0f){
        key[blockID] = true;
        hasNonZero = true;
        return;
      }
    }
    __syncthreads();
    if(!hasNonZero){
      key[blockID] = false;
      return;
    }
  }
}
__global__ void randInitMatrix(unsigned long size, float* mtx){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  if(globalID < size){
    mtx[globalID] = ((float)(clock64()%1000))/1000.0f;
  }
}
__global__ void multiplyMatrices(float *matrixA, float *matrixB, float *matrixC, long diffDimA, long comDim, long diffDimB){

  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  long currentIndex = globalID;

  if(currentIndex < (diffDimA * diffDimB)){

    long iIndex = currentIndex / diffDimB;
    long jIndex = currentIndex % diffDimB;

    float sum = 0;

    for(int k = 0; k < comDim; k++){

      sum += (matrixA[iIndex * comDim + k] * matrixB[k * diffDimB + jIndex]);
    }

    matrixC[iIndex * diffDimB + jIndex] = sum;
  }
}
__global__ void multiplyMatrices(float *matrixA, float *matrixB, uint32 *resultTranspose, long diffDimA, long comDim, long diffDimB){

  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  long currentIndex = globalID;

  if(currentIndex < (diffDimA * diffDimB)){

    long iIndex = currentIndex / diffDimB;
    long jIndex = currentIndex % diffDimB;

    float sum = 0;

    for(int k = 0; k < comDim; k++){

      sum += (matrixA[iIndex * comDim + k] * matrixB[k * diffDimB + jIndex]);
    }

    //result[iIndex * diffDimB + jIndex] = __float_as_uint(sum);
    if(sum == 0.0f) printf("YO\n");
    resultTranspose[jIndex * diffDimA + iIndex] = floatToOrderedInt(sum);

  }
}
void executeMultiplyMatrices(float *matrixA, float *matrixB, float* &matrixC, long diffDimA, long comDim, long diffDimB){

  float* matrixADevice, *matrixBDevice, *matrixCDevice;

  CudaSafeCall(hipMalloc((void**)&matrixADevice, diffDimA*comDim*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&matrixBDevice, comDim*diffDimB*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&matrixCDevice, diffDimA*diffDimB*sizeof(float)));

  CudaSafeCall(hipMemcpy(matrixADevice, matrixA, diffDimA*comDim*sizeof(float), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixBDevice, matrixB, comDim*diffDimB*sizeof(float), hipMemcpyHostToDevice));

  dim3 grid, block;

  getFlatGridBlock(diffDimA*diffDimB, grid, block);

  multiplyMatrices<<<grid, block>>>(matrixADevice, matrixBDevice, matrixCDevice, diffDimA, comDim, diffDimB);

  CudaSafeCall(hipMemcpy(matrixC, matrixCDevice, diffDimA*diffDimB*sizeof(float), hipMemcpyDeviceToHost));

  CudaSafeCall(hipFree(matrixADevice));
  CudaSafeCall(hipFree(matrixBDevice));
  CudaSafeCall(hipFree(matrixCDevice));

}

void getFlatGridBlock(unsigned long size, dim3 &grid, dim3 &block) {
  if(2147483647 > size){
    grid.x = size;
  }
  else if((unsigned long) 2147483647 * 1024 > size){
    grid.x = 2147483647;
    block.x = 1024;
    while(block.x * grid.x > size){
      block.x--;
    }
    block.x++;
  }
  else{
    grid.x = 65535;
    block.x = 1024;
    grid.y = 1;
    while(grid.x * grid.y * block.x < size){
      grid.y++;
    }
  }
}
void getGrid(unsigned long size, dim3 &grid, int blockSize) {
  if(2147483647 > size){
    grid.x = size;
  }
  else{
    grid.x = 65535;
    grid.y = 1;
    while(grid.x * grid.y * grid.y < size){
      grid.y++;
    }
  }
}
float* executeNormalization(uint32* mtx, unsigned long size){
  uint32 max = 0;
  uint32 min = UINT32_MAX;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(size, grid, block);

  float* norm = new float[size];
  uint32* maxd;
  uint32* mind;
  uint32* matrixDevice;
  float* normDevice;
  CudaSafeCall(hipMalloc((void**)&maxd, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&mind, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&matrixDevice, size*sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&normDevice, size*sizeof(float)));
  CudaSafeCall(hipMemcpy(maxd, &max, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(mind, &min, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, size*sizeof(uint32), hipMemcpyHostToDevice));

  std::cout<<"searching for max and min"<<std::endl;
  findMinMax<<<grid,block>>>(matrixDevice, size, mind, maxd);
  hipDeviceSynchronize();
  CudaCheckError();
  std::cout<<"executing normalization"<<std::endl;
  normalize<<<grid,block>>>(matrixDevice, normDevice, mind, maxd, size);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(&max, maxd, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(&min, mind, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(norm, normDevice, size*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(maxd));
  CudaSafeCall(hipFree(mind));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(normDevice));
  printf("whole video - (uint32) min = %d, max = %d\n",min,max);
  return norm;

}
bool* generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, unsigned long &numPixelsWithValues){
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  block.x = (numTimePoints < 1024) ? numTimePoints : 1024;
  getGrid(numPixels, grid, block.x);

  bool* key = new bool[numPixels];

  float* matrixDevice;
  bool* keyDevice;

  CudaSafeCall(hipMalloc((void**)&matrixDevice, numPixels*numTimePoints*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&keyDevice, numPixels*sizeof(float)));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, numPixels*numTimePoints*sizeof(float), hipMemcpyHostToDevice));
  std::cout<<"generating key to eradicate pixels that are always 0 = ";

  generateKey<<<grid,block>>>(numPixels, numTimePoints, matrixDevice, keyDevice);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(key, keyDevice, numPixels*sizeof(bool), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(keyDevice));
  for(int p = 0; p < numPixels; ++p){
    if(key[p]) ++numPixelsWithValues;
  }
  std::cout<<numPixels - numPixelsWithValues<<std::endl;

  return key;

}
float* minimizeVideo(unsigned long numPixels, unsigned long numPixelsWithValues, unsigned int numTimePoints, float* mtx, bool* key){
  std::cout<<"minimizing video due existence of all 0 rows"<<std::endl;
  float* minimizedVideo = new float[numPixelsWithValues*numTimePoints];
  int currentPixel = 0;
  for(int p = 0; p < numPixels; ++p){
    if(key[p]){
      memcpy(&minimizedVideo[currentPixel*numTimePoints], mtx + p*numTimePoints, numTimePoints*sizeof(float));
      ++currentPixel;
    }
  }
  return minimizedVideo;
}

void performNNMF(float* &W, float* &H, float* V, unsigned int k, unsigned long numPixels, unsigned int numTimePoints, std::string baseDir){
  clock_t nnmfTimer;
  nnmfTimer = clock();
  std::cout<<"starting nnmf"<<std::endl;
  std::cout<<"ensuring positivity"<<std::endl;
  float* dV;
  int* globalMin;
  int maxInt = INT_MAX;
  CudaSafeCall(hipMalloc((void**)&globalMin, sizeof(int)));
  CudaSafeCall(hipMemcpy(globalMin, &maxInt, sizeof(int), hipMemcpyHostToDevice));
  CudaSafeCall(hipMalloc((void**)&dV, numPixels*numTimePoints*sizeof(float)));
  CudaSafeCall(hipMemcpy(dV, V, numPixels*numTimePoints*sizeof(float), hipMemcpyHostToDevice));
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(numPixels*numTimePoints, grid, block);
  ensurePositivity<<<grid,block>>>(dV, numPixels*numTimePoints, globalMin);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(V, dV, numPixels*numTimePoints*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(dV));
  CudaSafeCall(hipFree(globalMin));

  std::cout<<"Preparing data for python"<<std::endl;

  npy_intp vdim[] = {numPixels, numTimePoints};
  npy_intp wdim[] = {numPixels, k};
  npy_intp hdim[] = {k, numTimePoints};

  /*
    NOW USE PYTHON TO EXECUTE NNMF WITH TENSORFLOW
  */

  //define python objects
  PyObject *pyV, *pyW, *pyH;
  PyObject *scalarK, *scalarTP, *scalarPix,*scalarIterations;
  PyObject *args;
  PyObject *whReturn;

  //launch python interpreter
  Py_Initialize();
  import_array1();
  if(!Py_IsInitialized()){
    std::cout<<"Error initializing embedded python handler"<<std::endl;
    exit(-1);
  }
  else{
    std::cout<<"Embedded python handler initialized"<<std::endl;
  }

  PyObject* syspath = PySys_GetObject("path");
  PyList_Append(syspath, PyUnicode_FromString("./src"));

  std::cout<<"loading python module"<<std::endl;
  PyObject* myModule = PyImport_ImportModule("tfNNMF");
  if(!myModule){
    std::cout<<"tfNNMF cannot be imported"<<std::endl;
    PyErr_Print();
    exit(-1);
  }
  PyObject* myFunction = PyObject_GetAttrString(myModule, "tensorflowNNMF");

  scalarK = PyLong_FromUnsignedLong(k);
  scalarPix = PyLong_FromUnsignedLong(numPixels);
  scalarTP = PyLong_FromUnsignedLong(numTimePoints);
  scalarIterations = PyLong_FromUnsignedLong(1000);

  std::cout<<"loading V matrix into numpy array"<<std::endl;
  pyV = PyArray_SimpleNew(2, vdim, NPY_FLOAT);
  float* npy = (float *) PyArray_DATA(reinterpret_cast<PyArrayObject*>(pyV));
  for(int i = 0; i < numPixels; ++i){
    memcpy(npy, V + (i*numTimePoints), sizeof(float)*numTimePoints);
    npy += numTimePoints;
  }
  delete[] V;

  args = PyTuple_New(3);
  PyTuple_SetItem(args, 0, pyV);
  PyTuple_SetItem(args, 1, scalarK);
  PyTuple_SetItem(args, 2, scalarIterations);

  whReturn = PyObject_CallObject(myFunction, args);
  if(!whReturn){
    std::cout<<"Error in execution of tfnnmf.py"<<std::endl;
    PyErr_Print();
    exit(-1);
  }

  pyW = PyTuple_GetItem(whReturn, 0);
  pyH = PyTuple_GetItem(whReturn, 1);

  float* tempW;
  float* tempH;

  tempW = (float *) PyArray_GETPTR1(reinterpret_cast<PyArrayObject*>(pyW), 0);
  tempH = (float *) PyArray_GETPTR1(reinterpret_cast<PyArrayObject*>(pyH), 0);
  for(int i = 0; i < numPixels*k; ++i){
    W[i] = tempW[i];
  }
  for(int i = 0;i < k*numTimePoints; ++i){
    H[i] = tempH[i];
  }

  Py_DECREF(syspath);
  Py_DECREF(myFunction);
  Py_DECREF(myModule);
  Py_DECREF(pyV);
  Py_DECREF(pyW);
  Py_DECREF(pyH);
  Py_DECREF(scalarK);
  Py_DECREF(scalarPix);
  Py_DECREF(scalarTP);
  Py_DECREF(scalarIterations);
  Py_Finalize();
}
