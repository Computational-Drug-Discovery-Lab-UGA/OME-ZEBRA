#include "hip/hip_runtime.h"
#include "cuda_zebra.cuh"

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  // err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

__global__ void findMinMax(uint32* mtx, unsigned long size, uint32* min, uint32* max){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  __shared__ uint32 bmax;
  __shared__ uint32 bmin;
  bmax = 0;
  bmin = UINT32_MAX;
  __syncthreads();
  if(globalID < size){
    uint32 value = mtx[globalID];
    if(value != 0){
      atomicMax(&bmax, value);
      atomicMin(&bmin, value);
    }
  }
  __syncthreads();
  if(threadIdx.x == 0){
    atomicMax(max, bmax);
    atomicMin(min, bmin);
  }
}
__global__ void normalize(uint32 *mtx, float *normals, uint32* min, uint32* max, unsigned long size) {
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float currentValue = 0;
  float dmin = static_cast<float>(*min);
  float dmax = static_cast<float>(*max);
  while(globalID < size){
    if (mtx[globalID] != 0) {
      currentValue = static_cast<float>(mtx[globalID]) - dmin;
      currentValue /= (dmax - dmin);
    }
    normals[globalID] = currentValue;
    normals[globalID] = 1.0f / (1.0f + expf((-10.0f * currentValue) + 7.5));
    //printf("%f\n",normals[globalID]);
    globalID += stride;
  }
}
__global__ void generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, bool* key){
  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numPixels){
    __shared__ bool hasNonZero;
    hasNonZero = false;
    __syncthreads();
    for(int tp = threadIdx.x; tp < numTimePoints; tp += blockDim.x){
      if(hasNonZero) return;
      if(mtx[blockID*numTimePoints + tp] != 0.0f){
        key[blockID] = true;
        hasNonZero = true;
        return;
      }
    }
    __syncthreads();
    if(!hasNonZero){
      key[blockID] = false;
      return;
    }
  }
}
__global__ void randInitMatrix(unsigned long size, float* mtx){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  if(globalID < size){
    mtx[globalID] = ((float)(clock64()%1000))/1000.0f;
  }
}
__global__ void multiplyMatrices(float *matrixA, float *matrixB, float *matrixC,
                                 long diffDimA, long comDim, long diffDimB){

  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  long currentIndex = globalID;

  if(currentIndex < (diffDimA * diffDimB)){

    long iIndex = currentIndex / diffDimB;
    long jIndex = currentIndex % diffDimB;

    float sum = 0;

    for(int k = 0; k < comDim; k++){

      sum += (matrixA[iIndex * comDim + k] * matrixB[k * diffDimB + jIndex]);
    }

    matrixC[iIndex * diffDimB + jIndex] = sum;
  }
}

void executeMultiplyMatrices(float *matrixA, float *matrixB, float* &matrixC,
                                 long diffDimA, long comDim, long diffDimB){

  float* matrixADevice, *matrixBDevice, *matrixCDevice;

  CudaSafeCall(hipMalloc((void**)&matrixADevice, diffDimA*comDim*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&matrixBDevice, comDim*diffDimB*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&matrixCDevice, diffDimA*diffDimB*sizeof(float)));

  CudaSafeCall(hipMemcpy(matrixADevice, matrixA, diffDimA*comDim*sizeof(float), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixBDevice, matrixB, comDim*diffDimB*sizeof(float), hipMemcpyHostToDevice));

  dim3 grid, block;

  getFlatGridBlock(diffDimA*diffDimB, grid, block);

  multiplyMatrices<<<grid, block>>>(matrixADevice, matrixBDevice, matrixCDevice, diffDimA, comDim, diffDimB);

  CudaSafeCall(hipMemcpy(matrixC, matrixCDevice, diffDimA*diffDimB*sizeof(float), hipMemcpyDeviceToHost));

  CudaSafeCall(hipFree(matrixADevice));
  CudaSafeCall(hipFree(matrixBDevice));
  CudaSafeCall(hipFree(matrixCDevice));

}

void getFlatGridBlock(unsigned long size, dim3 &grid, dim3 &block) {
  if(2147483647 > size){
    grid.x = size;
  }
  else if((unsigned long) 2147483647 * 1024 > size){
    grid.x = 2147483647;
    block.x = 1024;
    while(block.x * grid.x > size){
      block.x--;
    }
    block.x++;
  }
  else{
    grid.x = 65535;
    block.x = 1024;
    grid.y = 1;
    while(grid.x * grid.y * block.x < size){
      grid.y++;
    }
  }
}
void getGrid(unsigned long size, dim3 &grid, int blockSize) {
  if(2147483647 > size){
    grid.x = size;
  }
  else{
    grid.x = 65535;
    grid.y = 1;
    while(grid.x * grid.y * grid.y < size){
      grid.y++;
    }
  }
}
float* executeNormalization(uint32* mtx, unsigned long size){
  uint32 max = 0;
  uint32 min = UINT32_MAX;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(size, grid, block);

  float* norm = new float[size];
  uint32* maxd;
  uint32* mind;
  uint32* matrixDevice;
  float* normDevice;
  CudaSafeCall(hipMalloc((void**)&maxd, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&mind, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&matrixDevice, size*sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&normDevice, size*sizeof(float)));
  CudaSafeCall(hipMemcpy(maxd, &max, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(mind, &min, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, size*sizeof(uint32), hipMemcpyHostToDevice));

  std::cout<<"searching for max and min"<<std::endl;
  findMinMax<<<grid,block>>>(matrixDevice, size, mind, maxd);
  hipDeviceSynchronize();
  CudaCheckError();
  std::cout<<"executing normalization"<<std::endl;
  normalize<<<grid,block>>>(matrixDevice, normDevice, mind, maxd, size);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(&max, maxd, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(&min, mind, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(norm, normDevice, size*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(maxd));
  CudaSafeCall(hipFree(mind));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(normDevice));
  printf("whole video - (uint32) min = %d, max = %d\n",min,max);
  return norm;

}
bool* generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, unsigned long &numPixelsWithValues){
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  block.x = (numTimePoints < 1024) ? numTimePoints : 1024;
  getGrid(numPixels, grid, block.x);

  bool* key = new bool[numPixels];

  float* matrixDevice;
  bool* keyDevice;

  CudaSafeCall(hipMalloc((void**)&matrixDevice, numPixels*numTimePoints*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&keyDevice, numPixels*sizeof(float)));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, numPixels*numTimePoints*sizeof(float), hipMemcpyHostToDevice));
  std::cout<<"generating key to eradicate pixels that are always 0 = ";

  generateKey<<<grid,block>>>(numPixels, numTimePoints, matrixDevice, keyDevice);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(key, keyDevice, numPixels*sizeof(bool), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(keyDevice));
  for(int p = 0; p < numPixels; ++p){
    if(key[p]) ++numPixelsWithValues;
  }
  std::cout<<numPixels - numPixelsWithValues<<std::endl;

  return key;

}
float* minimizeVideo(unsigned long numPixels, unsigned long numPixelsWithValues, unsigned int numTimePoints, float* mtx, bool* key){
  std::cout<<"minimizing video due existence of all 0 rows"<<std::endl;
  float* minimizedVideo = new float[numPixelsWithValues*numTimePoints];
  int currentPixel = 0;
  for(int p = 0; p < numPixels; ++p){
    if(key[p]){
      memcpy(&minimizedVideo[currentPixel*numTimePoints], mtx + p*numTimePoints, numTimePoints*sizeof(float));
      ++currentPixel;
    }
  }
  return minimizedVideo;
}

void performNNMF(float* &W, float* &H, float* V, unsigned int k, unsigned long numPixels, unsigned int numTimePoints, std::string baseDir){
  clock_t nnmfTimer;
  nnmfTimer = clock();
  std::cout<<"starting nnmf"<<std::endl;
  float min = std::numeric_limits<float>::max();
  for(int i = 0; i < numPixels*numTimePoints; ++i){
    if(V[i] < min) min = V[i];
  }
  for(int i = 0; i < numPixels*numTimePoints; ++i){
    V[i] -= (min - .1);
  }
  /*WRITE NNMF.txt */
  std::string nmfFileName = baseDir + "NNMF.txt";
  std::ofstream NNMFile(nmfFileName);
  if(NNMFile.is_open()){
    for(int i = 0; i < numPixels*numTimePoints; ++i){
      if ((i + 1) % numTimePoints == 0) {
        NNMFile << V[i] << "\n";
      }
      else {
        NNMFile << V[i] << " ";
      }
    }
    NNMFile.close();
    std::cout<< nmfFileName <<" has been created.\n"<<std::endl;
  }
  else{
    std::cout<<"error cannot create"<< nmfFileName <<std::endl;
  }
  printf("writing NNMF.txt took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);
  nnmfTimer = clock();
  delete[] V;

  /*DO NMF*/
  std::string kS = std::to_string(k);
  pid_t pid = fork();
  int status;
  if(pid == 0){
    if(execl("bin/NMF_GPU","bin/NMF_GPU",nmfFileName.c_str(),"-k",kS.c_str(),"-j","10","-t","40","-i","20000", (char*)0) == -1){
      std::cout<<"ERROR CALLING NMF_GPU -> "<<strerror(errno)<<std::endl;
      exit(-1);
    }
  }
  else{
    while(-1 == wait(&status));
  }


  printf("nnmf took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);
  nnmfTimer = clock();
  W = new float[k*numPixels];
  H = new float[k*numTimePoints];
  std::cout<<"reading in h and w file"<<std::endl;
  std::string wFileName = nmfFileName + "_W.txt";
  std::string hFileName = nmfFileName + "_H.txt";
  std::cout<<"opening "<<wFileName<<" and"<<hFileName<<std::endl;
  std::string wLine = "";
  std::string hLine = "";
  std::ifstream wFile(wFileName);
  std::ifstream hFile(hFileName);
  std::istringstream hh;
  std::istringstream ww;
  if(wFile.is_open() && hFile.is_open()){
    for(int row = 0; row < numPixels; ++row){
      wLine = "";
      hLine = "";
      if(row < k){
        getline(hFile, hLine);
        hh = std::istringstream(hLine);
      }
      getline(wFile, wLine);
      ww = std::istringstream(wLine);
      for(int col = 0; col < k; ++col){
        ww >> W[row*k + col];
      }
      for(int col = 0; row < k && col < numTimePoints; ++col){
        hh >> H[row*numTimePoints + col];
      }
    }
    wFile.close();
    hFile.close();
  }
  else{
    std::cout<<"error cannot open W or H file"<<std::endl;
  }
  printf("reading h and w took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);

}
