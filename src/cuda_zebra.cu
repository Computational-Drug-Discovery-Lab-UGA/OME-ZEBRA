#include "hip/hip_runtime.h"
#include "cuda_zebra.cuh"

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  // err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

__global__ void findMinMax(uint32* mtx, unsigned long size, uint32* min, uint32* max){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  __shared__ uint32 bmax;
  __shared__ uint32 bmin;
  bmax = 0;
  bmin = UINT32_MAX;
  __syncthreads();
  if(globalID < size){
    uint32 value = mtx[globalID];
    if(value != 0){
      atomicMax(&bmax, value);
      atomicMin(&bmin, value);
    }
  }
  __syncthreads();
  if(threadIdx.x == 0){
    atomicMax(max, bmax);
    atomicMin(min, bmin);
  }
}
__global__ void normalize(uint32 *mtx, float *normals, uint32* min, uint32* max, unsigned long size) {
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  int stride = gridDim.x * gridDim.y * blockDim.x;
  float currentValue = 0;
  float dmin = static_cast<float>(*min);
  float dmax = static_cast<float>(*max);
  while(globalID < size){
    if (mtx[globalID] != 0) {
      currentValue = static_cast<float>(mtx[globalID]) - dmin;
      currentValue /= (dmax - dmin);
    }
    normals[globalID] = currentValue;
    normals[globalID] = 1.0f / (1.0f + expf((-10.0f * currentValue) + 7.5));
    //printf("%f\n",normals[globalID]);
    globalID += stride;
  }
}
__global__ void generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, bool* key){
  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numPixels){
    __shared__ bool hasNonZero;
    hasNonZero = false;
    __syncthreads();
    for(int tp = threadIdx.x; tp < numTimePoints; tp += blockDim.x){
      if(hasNonZero) return;
      if(mtx[blockID*numTimePoints + tp] != 0.0f){
        key[blockID] = true;
        hasNonZero = true;
        return;
      }
    }
    __syncthreads();
    if(!hasNonZero){
      key[blockID] = false;
      return;
    }
  }
}
__global__ void randInitMatrix(unsigned long size, float* mtx){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  if(globalID < size){
    mtx[globalID] = ((float)(clock64()%1000))/1000.0f;
  }
}
__global__ void multiplyMatrices(float *matrixA, float *matrixB, float *matrixC,
                                 long diffDimA, long comDim, long diffDimB){

  long blockID = blockIdx.y * gridDim.x + blockIdx.x;
  long globalID = blockID * blockDim.x + threadIdx.x;
  long currentIndex = globalID;

  if(currentIndex < (diffDimA * diffDimB)){

    long iIndex = currentIndex / diffDimB;
    long jIndex = currentIndex % diffDimB;

    float sum = 0;

    for(int k = 0; k < comDim; k++){

      sum += (matrixA[iIndex * comDim + k] * matrixB[k * diffDimB + jIndex]);
    }

    matrixC[iIndex * diffDimB + jIndex] = sum;
  }
}

void executeMultiplyMatrices(float *matrixA, float *matrixB, float* &matrixC,
                                 long diffDimA, long comDim, long diffDimB){

  float* matrixADevice, *matrixBDevice, *matrixCDevice;

  CudaSafeCall(hipMalloc((void**)&matrixADevice, diffDimA*comDim*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&matrixBDevice, comDim*diffDimB*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&matrixCDevice, diffDimA*diffDimB*sizeof(float)));

  CudaSafeCall(hipMemcpy(matrixADevice, matrixA, diffDimA*comDim*sizeof(float), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixBDevice, matrixB, comDim*diffDimB*sizeof(float), hipMemcpyHostToDevice));

  dim3 grid, block;

  getFlatGridBlock(diffDimA*diffDimB, grid, block);

  multiplyMatrices<<<grid, block>>>(matrixADevice, matrixBDevice, matrixCDevice, diffDimA, comDim, diffDimB);

  CudaSafeCall(hipMemcpy(matrixC, matrixCDevice, diffDimA*diffDimB*sizeof(float), hipMemcpyDeviceToHost));

  CudaSafeCall(hipFree(matrixADevice));
  CudaSafeCall(hipFree(matrixBDevice));
  CudaSafeCall(hipFree(matrixCDevice));

}

void getFlatGridBlock(unsigned long size, dim3 &grid, dim3 &block) {
  if(2147483647 > size){
    grid.x = size;
  }
  else if((unsigned long) 2147483647 * 1024 > size){
    grid.x = 2147483647;
    block.x = 1024;
    while(block.x * grid.x > size){
      block.x--;
    }
    block.x++;
  }
  else{
    grid.x = 65535;
    block.x = 1024;
    grid.y = 1;
    while(grid.x * grid.y * block.x < size){
      grid.y++;
    }
  }
}
void getGrid(unsigned long size, dim3 &grid, int blockSize) {
  if(2147483647 > size){
    grid.x = size;
  }
  else{
    grid.x = 65535;
    grid.y = 1;
    while(grid.x * grid.y * grid.y < size){
      grid.y++;
    }
  }
}
float* executeNormalization(uint32* mtx, unsigned long size){
  uint32 max = 0;
  uint32 min = UINT32_MAX;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(size, grid, block);

  float* norm = new float[size];
  uint32* maxd;
  uint32* mind;
  uint32* matrixDevice;
  float* normDevice;
  CudaSafeCall(hipMalloc((void**)&maxd, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&mind, sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&matrixDevice, size*sizeof(uint32)));
  CudaSafeCall(hipMalloc((void**)&normDevice, size*sizeof(float)));
  CudaSafeCall(hipMemcpy(maxd, &max, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(mind, &min, sizeof(uint32), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, size*sizeof(uint32), hipMemcpyHostToDevice));

  std::cout<<"searching for max and min"<<std::endl;
  findMinMax<<<grid,block>>>(matrixDevice, size, mind, maxd);
  hipDeviceSynchronize();
  CudaCheckError();
  std::cout<<"executing normalization"<<std::endl;
  normalize<<<grid,block>>>(matrixDevice, normDevice, mind, maxd, size);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(&max, maxd, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(&min, mind, sizeof(uint32), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(norm, normDevice, size*sizeof(float), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(maxd));
  CudaSafeCall(hipFree(mind));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(normDevice));
  printf("whole video - (uint32) min = %d, max = %d\n",min,max);
  return norm;

}
bool* generateKey(unsigned long numPixels, unsigned int numTimePoints, float* mtx, unsigned long &numPixelsWithValues){
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  block.x = (numTimePoints < 1024) ? numTimePoints : 1024;
  getGrid(numPixels, grid, block.x);

  bool* key = new bool[numPixels];

  float* matrixDevice;
  bool* keyDevice;

  CudaSafeCall(hipMalloc((void**)&matrixDevice, numPixels*numTimePoints*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&keyDevice, numPixels*sizeof(float)));
  CudaSafeCall(hipMemcpy(matrixDevice, mtx, numPixels*numTimePoints*sizeof(float), hipMemcpyHostToDevice));
  std::cout<<"generating key to eradicate pixels that are always 0 = ";

  generateKey<<<grid,block>>>(numPixels, numTimePoints, matrixDevice, keyDevice);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(key, keyDevice, numPixels*sizeof(bool), hipMemcpyDeviceToHost));
  CudaSafeCall(hipFree(matrixDevice));
  CudaSafeCall(hipFree(keyDevice));
  for(int p = 0; p < numPixels; ++p){
    if(key[p]) ++numPixelsWithValues;
  }
  std::cout<<numPixels - numPixelsWithValues<<std::endl;

  return key;

}
float* minimizeVideo(unsigned long numPixels, unsigned long numPixelsWithValues, unsigned int numTimePoints, float* mtx, bool* key){
  std::cout<<"minimizing video due existence of all 0 rows"<<std::endl;
  float* minimizedVideo = new float[numPixelsWithValues*numTimePoints];
  int currentPixel = 0;
  for(int p = 0; p < numPixels; ++p){
    if(key[p]){
      memcpy(&minimizedVideo[currentPixel*numTimePoints], mtx + p*numTimePoints, numTimePoints*sizeof(float));
      ++currentPixel;
    }
  }
  return minimizedVideo;
}


void performSVD(unsigned int numSigFig, unsigned long mValue, unsigned long nValue, float* originalMatrix, float* &sMatrix, float* &uMatrix, float* &vtMatrix) {
  std::cout<<"starting SVD"<<std::endl;
  magma_init(); // initialize Magma
  real_Double_t gpu_time, cpu_time;

  magma_int_t m=mValue, n=nValue, n2=m*n, min_mn = (((m)<(n))?(m):(n));
  printf("m=%d,n=%d\n",m,n);
  float *r; // a,r - mxn matrices
  float *u, *vt;// u - mxm matrix , vt - nxn matrix on the host
  float *s1; // vectors of singular values
  magma_int_t info;
  magma_int_t ione = 1;
  float work[1], error = 1.; // used in difference computations
  float mone = -1.0, * h_work ; // h_work - workspace
  float dummy[1];
  magma_int_t lwork ; // workspace size
  magma_int_t ISEED[4] = {0 ,0 ,0 ,1}; // seed

  // Allocate host memory
  magma_smalloc_cpu(&vt, n*n); // host memory for vt
  magma_smalloc_cpu(&u, m*n); // host memory for u
  magma_smalloc_cpu(&s1 , min_mn ); // host memory for s1
  magma_smalloc_pinned(&r,n2 ); // host memory for r
  magma_int_t nb = magma_get_sgesvd_nb(m,n); // optim . block size

  magma_sgesvd(MagmaSomeVec, MagmaSomeVec, m, n,
                NULL, m, NULL, NULL, m, NULL, n, dummy, -1,
                &info );

  lwork = (magma_int_t) MAGMA_S_REAL( dummy[0] );
  magma_smalloc_pinned(& h_work , lwork ); // host mem . for h_work



  // lapackf77_slarnv(&ione, ISEED, &n2, a);
  lapackf77_slacpy(MagmaFullStr, &m, &n, originalMatrix, &m, r, &m);

  // MAGMA
  gpu_time = magma_wtime();

  // compute the singular value decomposition of r ( copy of a)
  // and optionally the left and right singular vectors :
  // r = u* sigma *vt; the diagonal elements of sigma (s1 array )
  // are the singular values of a in descending order
  // the first min (m,n) columns of u contain the left sing . vec .
  // the first min (m,n) columns of vt contain the right sing .vec .
  magma_sgesvd(MagmaSomeVec,MagmaSomeVec,m,n,r,m,s1,u,m,vt,n,h_work,
  lwork,&info );

  gpu_time = magma_wtime() - gpu_time ;
  printf(" sgesvd gpu time: %7.5f\n", gpu_time); // Magma time
  std::cout<<"starting minimization with num singular values being "<<numSigFig<<std::endl;
  for (long i = 0; i < numSigFig; i++) {
    for (long j = 0; j < numSigFig; j++) {
      if (j == i) {
        sMatrix[i * numSigFig + j] = s1[i];
      }
      else {
        sMatrix[i * numSigFig + j] = 0.0f;
      }
    }
  }

  std::cout<<"sMatrix minimized = "<<numSigFig<<"x"<<numSigFig<<std::endl;

  for(long i = 0; i < m; i++){
    for(int j = 0; j < numSigFig; j++) {
        uMatrix[i*n + j] = u[i*n + j];
    }
  }

  std::cout<<"uMatrix minimized = "<<m<<"x"<<numSigFig<<std::endl;

  for(long i = 0; i < numSigFig; i++) {

    for(int j = 0; j < n; j++) {

        vtMatrix[i*n + j] = vt[i*n + j];

    }

  }

  std::cout<<"vtMatrix minimized = "<<numSigFig<<"x"<<n<<std::endl;


  // values
  // Free memory
  free(vt); // free host memory
  free(s1); //free host memory
  free(u); // free host memory
  magma_free_pinned( h_work ); // free host memory
  magma_free_pinned(r); // free host memory
  magma_finalize( ); // finalize Magma

}

void performNNMF(float* &W, float* &H, float* V, unsigned int k, unsigned long numPixels, unsigned int numTimePoints, std::string baseDir, bool svd){
  float* svdProduct;
  unsigned int numSigFig;
  float* sMatrix;
  float* uMatrix;
  float* vtMatrix;
  float* tempMatrix;
  if(svd){
    numSigFig = 200;
    /*DO SVD*/
    sMatrix = new float[numSigFig*numSigFig];
    uMatrix = new float[numPixels*numSigFig];
    vtMatrix = new float[numSigFig*numTimePoints];
    performSVD(numSigFig, numPixels, numTimePoints, V, sMatrix, uMatrix, vtMatrix);
    delete[] V;
    tempMatrix = new float[numPixels*numSigFig];
    executeMultiplyMatrices(uMatrix, sMatrix, tempMatrix, numPixels, numSigFig, numSigFig);
    delete[] uMatrix;
    delete[] sMatrix;
    svdProduct = new float[numPixels*numTimePoints];
    executeMultiplyMatrices(tempMatrix, vtMatrix, svdProduct, numPixels, numSigFig, numTimePoints);
    delete[] vtMatrix;
    delete[] tempMatrix;
    numSigFig = 50;
    sMatrix = new float[numSigFig*numSigFig];
    uMatrix = new float[numPixels*numSigFig];
    vtMatrix = new float[numSigFig*numTimePoints];
    performSVD(numSigFig, numPixels, numTimePoints, svdProduct, sMatrix, uMatrix, vtMatrix);
    delete[] svdProduct;
    tempMatrix = new float[numPixels*numSigFig];
    executeMultiplyMatrices(uMatrix, sMatrix, tempMatrix, numPixels, numSigFig, numSigFig);
    delete[] uMatrix;
    delete[] sMatrix;
    svdProduct = new float[numPixels*numTimePoints];
    executeMultiplyMatrices(tempMatrix, vtMatrix, svdProduct, numPixels, numSigFig, numTimePoints);
    delete[] vtMatrix;
    delete[] tempMatrix;
  }
  else{
    svdProduct = V;
  }

  clock_t nnmfTimer;
  nnmfTimer = clock();
  std::cout<<"starting nnmf"<<std::endl;
  float min = std::numeric_limits<float>::max();
  for(int i = 0; i < numPixels*numTimePoints; ++i){
    if(svdProduct[i] < min) min = svdProduct[i];
  }
  for(int i = 0; i < numPixels*numTimePoints; ++i){
    svdProduct[i] -= (min - .1);
  }
  /*WRITE NNMF.txt */
  std::string nmfFileName = baseDir + "NNMF.txt";
  std::ofstream NNMFile(nmfFileName);
  if(NNMFile.is_open()){
    for(int i = 0; i < numPixels*numTimePoints; ++i){
      if ((i + 1) % numTimePoints == 0) {
        NNMFile << svdProduct[i] << "\n";
      }
      else {
        NNMFile << svdProduct[i] << " ";
      }
    }
    NNMFile.close();
    std::cout<< nmfFileName <<" has been created.\n"<<std::endl;
  }
  else{
    std::cout<<"error cannot create"<< nmfFileName <<std::endl;
  }
  printf("writing NNMF.txt took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);
  nnmfTimer = clock();
  delete[] svdProduct;

  /*DO NMF*/
  std::string executableLine = "./bin/NMF_GPU " + baseDir + "NNMF.txt -k " + std::to_string(k) + " -j 10 -t 40 -i 20000";
  std::system(executableLine.c_str());


  printf("nnmf took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);
  nnmfTimer = clock();
  W = new float[k*numPixels];
  H = new float[k*numTimePoints];
  std::cout<<"reading in h and w file"<<std::endl;
  std::string wFileName = nmfFileName + "_W.txt";
  std::string hFileName = nmfFileName + "_H.txt";
  std::cout<<"opening "<<wFileName<<" and"<<hFileName<<std::endl;
  std::string wLine = "";
  std::string hLine = "";
  std::ifstream wFile(wFileName);
  std::ifstream hFile(hFileName);
  std::istringstream hh;
  std::istringstream ww;
  if(wFile.is_open() && hFile.is_open()){
    for(int row = 0; row < numPixels; ++row){
      wLine = "";
      hLine = "";
      if(row < k){
        getline(hFile, hLine);
        hh = std::istringstream(hLine);
      }
      getline(wFile, wLine);
      ww = std::istringstream(wLine);
      for(int col = 0; col < k; ++col){
        ww >> W[row*k + col];
      }
      for(int col = 0; row < k && col < numTimePoints; ++col){
        hh >> H[row*numTimePoints + col];
      }
    }
    wFile.close();
    hFile.close();
  }
  else{
    std::cout<<"error cannot open W or H file"<<std::endl;
  }
  printf("reading h and w took %f seconds.\n\n", ((float) clock() - nnmfTimer)/CLOCKS_PER_SEC);

}
